#include "hip/hip_runtime.h"
﻿#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
//device parameter//
#define  ndim	    2
#define  ip         2
#define  kp         3
#define  DPN        3  //device per note
#define  stepall    100
#define  iprint     10
#define  idata_3d   100000
//droplet parameter//
#define  nx         120
#define  ny         120
#define  nz         120
#define  radd       30.0 //1不用調
#define  thick      5.0
#define  tau_h      0.5
#define  tau_l      0.05
#define  tau_g      0.5
#define  rho_l      1.0
#define  rho_g      0.001
#define  sigma      0.001
#define  bo         100.0

//0 one bubble rising ,1 two bubble rising
#define  condition  0

//condition 1
#define  distance_x 0.0
#define  distance_z 10.0
#define  radd_t     30.0
#define  radd_b     30.0

//constant parameter//
#define  thita      10
#define  dx         1.0
#define  dt         1.0
#define  q          19

__constant__ double eex[q];
__constant__ double eey[q];
__constant__ double eez[q];
__constant__ double wwt[q];
__constant__ int    eet[q];

void parameter (double *beta,double *zeta,double *mobi,double *kappa,double *phic,double *gravity,double *ex_h,double *ey_h,double *ez_h,double *wt_h,int *et_h)
{
	*zeta =(double)thick*dx;
	*beta =(double)12.0*sigma/(*zeta);
	*kappa=(double)(*beta)*(*zeta)*(*zeta)/8.0;
	*mobi =(double)0.02/(*beta);
	double omega=-cos(thita*M_PI/180.0);
	*phic =omega*pow(2.0*(*kappa)*(*beta),0.5);
	*gravity = bo*sigma/(rho_l-rho_g)/(2*radd)/(2*radd);
	//ex
	ex_h[ 0]= 0.0;
	ex_h[ 1]= 1.0;
	ex_h[ 2]=-1.0;
	ex_h[ 3]= 0.0;
	ex_h[ 4]= 0.0;
	ex_h[ 5]= 0.0;
	ex_h[ 6]= 0.0;
	ex_h[ 7]= 1.0;
	ex_h[ 8]=-1.0;
	ex_h[ 9]= 1.0;
	ex_h[10]=-1.0;
	ex_h[11]= 1.0;
	ex_h[12]=-1.0;
	ex_h[13]=-1.0;
	ex_h[14]= 1.0;
	ex_h[15]= 0.0;
	ex_h[16]= 0.0;
	ex_h[17]= 0.0;
	ex_h[18]= 0.0;
	//ey
	ey_h[ 0]= 0.0;
	ey_h[ 1]= 0.0;
	ey_h[ 2]= 0.0;
	ey_h[ 3]= 1.0;
	ey_h[ 4]=-1.0;
	ey_h[ 5]= 0.0;
	ey_h[ 6]= 0.0;
	ey_h[ 7]= 1.0;
	ey_h[ 8]=-1.0;
	ey_h[ 9]=-1.0;
	ey_h[10]= 1.0;
	ey_h[11]= 0.0;
	ey_h[12]= 0.0;
	ey_h[13]= 0.0;
	ey_h[14]= 0.0;
	ey_h[15]= 1.0;
	ey_h[16]=-1.0;
	ey_h[17]= 1.0;
	ey_h[18]=-1.0;
	//ez
	ez_h[ 0]= 0.0;
	ez_h[ 1]= 0.0;
	ez_h[ 2]= 0.0;
	ez_h[ 3]= 0.0;
	ez_h[ 4]= 0.0;
	ez_h[ 5]= 1.0;
	ez_h[ 6]=-1.0;
	ez_h[ 7]= 0.0;
	ez_h[ 8]= 0.0;
	ez_h[ 9]= 0.0;
	ez_h[10]= 0.0;
	ez_h[11]= 1.0;
	ez_h[12]=-1.0;
	ez_h[13]= 1.0;
	ez_h[14]=-1.0;
	ez_h[15]= 1.0;
	ez_h[16]=-1.0;
	ez_h[17]=-1.0;
	ez_h[18]= 1.0;
	//wt
	wt_h[ 0]=1.0/ 3.0;
	wt_h[ 1]=1.0/18.0;
	wt_h[ 2]=1.0/18.0;
	wt_h[ 3]=1.0/18.0;
	wt_h[ 4]=1.0/18.0;
	wt_h[ 5]=1.0/18.0;
	wt_h[ 6]=1.0/18.0;
	wt_h[ 7]=1.0/36.0;
	wt_h[ 8]=1.0/36.0;
	wt_h[ 9]=1.0/36.0;
	wt_h[10]=1.0/36.0;
	wt_h[11]=1.0/36.0;
	wt_h[12]=1.0/36.0;
	wt_h[13]=1.0/36.0;
	wt_h[14]=1.0/36.0;
	wt_h[15]=1.0/36.0;
	wt_h[16]=1.0/36.0;
	wt_h[17]=1.0/36.0;
	wt_h[18]=1.0/36.0;
	int l;
	for(l=0;l<q;l++)
	{
	et_h[l]=(nx/ip+4)*((ny+4)*(int)ez_h[l]+(int)ey_h[l])+(int)ex_h[l];
	}
}

void initial_macro (double *c,double *m,double *b,double *p,double *u,double *v,double *w)
{
	int i,j,k,index;
	double icent,jcent,kcent;
	for(i=0;i<nx;i++){
	for(j=0;j<ny;j++){
	for(k=0;k<nz;k++){
	index=nx*(k*ny+j)+i;
	c[index]=0.0;
	m[index]=0.0;
	b[index]=0.0;
	p[index]=0.0;
	u[index]=0.0;
	v[index]=0.0;
	w[index]=0.0;
	}}}
	
	icent=(double)(nx-1.0)/2.0;
	jcent=(double)(ny-1.0)/2.0;
	kcent=(double)(nz-1.0)/2.0;
	
	if(condition==1){
	double icent_r=icent+0.5*distance_x;
	double icent_l=icent-0.5*distance_x;
	double kcent_b=50;
	double kcent_t=kcent_b+thick+(radd_t+radd_b)+distance_z;
	int    mid    =0.5*(distance_z+thick)+50+radd_b;
	double raddd=radd+thick/2.0+1.0;
	
	for(i=0;i<nx;i++){
	for(j=0;j<ny;j++){
	for(k=0;k<mid;k++){
	double rad=sqrt( (i-icent_l)*(i-icent_l)+(j-jcent)*(j-jcent)+(k-kcent_b)*(k-kcent_b));
	index=nx*(k*ny+j)+i;
	c[index]=(double)0.5-(double)0.5*tanh(2.0*(radd_b-rad)/thick);
	}}}
	
 	for(i=0;i<nx;i++){
	for(j=0;j<ny;j++){
	for(k=mid;k<nz;k++){
	double rad=sqrt( (i-icent_r)*(i-icent_r)+(j-jcent)*(j-jcent)+(k-kcent_t)*(k-kcent_t));
	index=nx*(k*ny+j)+i;
	c[index]=(double)0.5-(double)0.5*tanh(2.0*(radd_t-rad)/thick);
	}}}
	}
	
	else{
	for(i=0;i<nx;i++){
	for(j=0;j<ny;j++){
	for(k=0;k<nz;k++){
	double rad=sqrt( (i-icent)*(i-icent)+(j-jcent)*(j-jcent)+(k-kcent)*(k-kcent));
	index=nx*(k*ny+j)+i;
	c[index]=(double)0.5-(double)0.5*tanh(2.0*(radd-rad)/thick);
	}}}}
}

void array_2D_do (double *phi,double *phi_do)
{
	int i,j,k,index;
	int ii,jj,kk,iindex;
	int iside;
	int xd=nx/ip;
	jj=-1;
	kk=0;
	iside=0;
	
	for(k=0;k<nz;k++){
	for(j=0;j<ny;j++){
	for(i=0;i<nx;i++){
	index=nx*(k*ny+j)+i;
	ii=i%xd;
	if(ii == 0){
	jj=jj+1;
	}
	if(jj == ny){
	kk=kk+1;
	jj=0;
	}
	if(kk == nz){
	iside=iside+1;
	kk=0;
	}
	
	ii=ii+xd*iside;
	iindex=nx*(kk*ny+jj)+ii;
	phi_do[index]=phi[iindex];
}
}
}
}

void array_2D_undo (double *phi,double *phi_do)
{
	int i,j,k,index;
	int ii,jj,kk,iindex;
	int iside;
	int xd=nx/ip;
	jj=-1;
	kk=0;
	iside=0;
	
	for(k=0;k<nz;k++){
	for(j=0;j<ny;j++){
	for(i=0;i<nx;i++){
	index=nx*(k*ny+j)+i;
	ii=i%xd;
	if(ii == 0){
	jj=jj+1;
	}
	if(jj == ny){
	kk=kk+1;
	jj=0;
	}
	if(kk == nz){
	iside=iside+1;
	kk=0;
	}
	
	ii=ii+xd*iside;
	iindex=nx*(kk*ny+jj)+ii;
	phi[iindex]=phi_do[index];
}
}
}
}

void array_1D_undo (double *phi,double *phi_do)
{
	int i,k,index;
	int ii,kk,iindex;
	int iside;
	int xd=nx/ip;
	kk=0;
	iside=0;
	
	for(k=0;k<nz;k++){
	for(i=0;i<nx;i++){
	index=nx*k+i;
	ii=i%xd;
	if(ii == 0){
	kk=kk+1;
	}
	if(kk == nz){
	iside=iside+1;
	kk=0;
	}
	
	ii=ii+xd*iside;
	iindex=nx*kk+ii;
	phi[iindex]=phi_do[index];
}
}
}

__device__ int index_3d (int i, int j,int k)
{
	int ans=(nx/ip+4)*((ny+4)*k+j)+i;
	return ans;
}

__device__ int index_3d_x (int i, int j,int k)
{
	int ans=(ny+4)*((nz/kp+4)*i+k)+j;
	return ans;
}

__device__ int index_4d (int i, int j,int k,int l)
{
	int ans=(nx/ip+4)*((ny+4)*((nz/kp+4)*l+k)+j)+i;
	return ans;
}

__global__ void array_do( double *phi_d, double *phi)
{
	int ii=threadIdx.x;
	int jj= blockIdx.x%ny;
	int kk= blockIdx.x/ny;
	int iindex	=(nx/ip)*(kk*ny+jj)+ii;
	
	int i=threadIdx.x+2;
	int j= blockIdx.x%ny+2;
	int k= blockIdx.x/ny+2;
	int index=index_3d(i,j,k);
	phi[index]=phi_d[iindex];
}
__global__ void array_undo( double *phi_d, double *phi)
{
	int ii=threadIdx.x;
	int jj= blockIdx.x%ny;
	int kk= blockIdx.x/ny;
	int iindex	=(nx/ip)*(kk*ny+jj)+ii;
	
	int i=threadIdx.x+2;
	int j= blockIdx.x%ny+2;
	int k= blockIdx.x/ny+2;
	int index=index_3d(i,j,k);
	phi_d[iindex]=phi[index];
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                boundary                                                        //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void boundary_ym( double *phi)
{
	int i= blockIdx.x;
	int k=threadIdx.x;
	int distance=(ny)*(nx/ip+4);
	
	for(int j=0;j<2;j++){
	int index	=index_3d(i,j,k);
	phi[index]=phi[index+distance];
	}
	for(int j=ny+2;j<ny+4;j++){
	int index	=index_3d(i,j,k);
	phi[index]=phi[index-distance];
	}
}

__global__ void boundary_zm1( double *phi, double *t_phi )
{
	int k,index,index_t;
	int i= blockIdx.x;
	int j=threadIdx.x;
	k=2;
	index	=index_3d(i,j,k);
	index_t	=index_3d(i,j,1);
	t_phi[index_t]=phi[index];

	k=nz/kp+1;
	index	=index_3d(i,j,k);
	index_t	=index_3d(i,j,2);
	t_phi[index_t]=phi[index];
}

__global__ void boundary_xm1( double *phi, double *t_phi  )
{
	int i,index,index_t;
	int j= blockIdx.x;
	int k=threadIdx.x;
	i=2;
	index	=index_3d(i,j,k);
	index_t	=index_3d_x(1,j,k);
	t_phi[index_t]=phi[index];

	i=nx/ip+1;
	index	=index_3d(i,j,k);
	index_t	=index_3d_x(2,j,k);
	t_phi[index_t]=phi[index];
}

__global__ void boundary_zm1_undo( double *phi, double *t_phi)
{
	int k,index,index_t;
	int i= blockIdx.x;
	int j=threadIdx.x;
	k=1;
	index	=index_3d(i,j,k);
	index_t	=index_3d(i,j,0);
	phi[index]=t_phi[index_t];

	k=nz/kp+2;
	index	=index_3d(i,j,k);
	index_t	=index_3d(i,j,3);
	phi[index]=t_phi[index_t];
}

__global__ void boundary_xm1_undo( double *phi, double *t_phi)
{
	int i,index,index_t;
	int j= blockIdx.x;
	int k=threadIdx.x;
	i=1;
	index	=index_3d(i,j,k);
	index_t	=index_3d_x(0,j,k);
	phi[index]=t_phi[index_t];

	i=nx/ip+2;
	index	=index_3d(i,j,k);
	index_t	=index_3d_x(3,j,k);
	phi[index]=t_phi[index_t];
}

__global__ void boundary_zm2( double *phi, double *t_phi )
{
	int k,l,index,index_t;
	int i= blockIdx.x;
	int j=threadIdx.x;
	
	for(l=0;l<2;l++){
	k=2;
	index	=index_3d(i,j,k+l);
	index_t	=index_3d(i,j,2+l);
	t_phi[index_t]=phi[index];
	
	k=nz/kp;
	index	=index_3d(i,j,k+l);
	index_t	=index_3d(i,j,4+l);
	t_phi[index_t]=phi[index];
	}
}

__global__ void boundary_xm2( double *phi, double *t_phi )
{
	int i,l,index,index_t;
	int j= blockIdx.x;
	int k=threadIdx.x;
	
	for(l=0;l<2;l++){
	i=2;
	index	=index_3d(i+l,j,k);
	index_t	=index_3d_x(2+l,j,k);
	t_phi[index_t]=phi[index];
	
	i=nx/ip;
	index	=index_3d(i+l,j,k);
	index_t	=index_3d_x(4+l,j,k);
	t_phi[index_t]=phi[index];
	}
}

__global__ void boundary_zm2_undo( double *phi, double *t_phi)
{
	int k,l,index,index_t;
	int i= blockIdx.x;
	int j=threadIdx.x;
	
	for(l=0;l<2;l++){
	k=0;
	index	=index_3d(i,j,k+l);
	index_t	=index_3d(i,j,0+l);
	phi[index]=t_phi[index_t];
	
	k=nz/kp+2;
	index	=index_3d(i,j,k+l);
	index_t	=index_3d(i,j,6+l);
	phi[index]=t_phi[index_t];
	}
}

__global__ void boundary_xm2_undo( double *phi, double *t_phi)
{
	int i,l,index,index_t;
	int j= blockIdx.x;
	int k=threadIdx.x;
	
	for(l=0;l<2;l++){
	i=0;
	index	=index_3d(i+l,j,k);
	index_t	=index_3d_x(0+l,j,k);
	phi[index]=t_phi[index_t];
	
	i=nx/ip+2;
	index	=index_3d(i+l,j,k);
	index_t	=index_3d_x(6+l,j,k);
	phi[index]=t_phi[index_t];
	}
}

__global__ void boundary_yd_bc( double *g,double *h)
{
	int i= blockIdx.x+2;
	int j,index_l;
	int zd=nz/kp;
	int l=threadIdx.x;
	int distance=(ny)*(nx/ip+4);

	for(int k=2;k<zd+2;k=k+zd-1){
	j=1;
	index_l=index_4d(i,j,k,l);
	g[index_l]=g[index_l+distance];
	h[index_l]=h[index_l+distance];
	j=ny+2;
	index_l=index_4d(i,j,k,l);
	g[index_l]=g[index_l-distance];
	h[index_l]=h[index_l-distance];
	}
}

__global__ void boundary_yd_bc_x( double *g,double *h)
{
	int k= blockIdx.x+2;
	int j,index_l;
	int xd=nx/ip;
	int l=threadIdx.x;
	int distance=(ny)*(nx/ip+4);
	
	for(int i=2;i<xd+2;i=i+xd-1){
	j=1;
	index_l=index_4d(i,j,k,l);
	g[index_l]=g[index_l+distance];
	h[index_l]=h[index_l+distance];
	j=ny+2;
	index_l=index_4d(i,j,k,l);
	g[index_l]=g[index_l-distance];
	h[index_l]=h[index_l-distance];
	}
}

__global__ void boundary_zd( double *phi,double *t_phi )
{
	int i= blockIdx.x+1;
	int j=threadIdx.x+1;
	int k,index_l,index_l_t;
	int xd=nx/ip;
	int l_top[5]={5,11,13,15,18};
	int l_bot[5]={6,12,14,16,17};

	for(int l=0;l<5;l++){
	k=2;
	index_l  =index_4d(i,j,k,l_bot[l]);
	index_l_t=((xd+4)*(1*(ny+4)+j)+i)*5+l;//k=1;q=5
	t_phi[index_l_t]=phi[index_l];
	
	k=nz/kp+1;
	index_l  =index_4d(i,j,k,l_top[l]);
	index_l_t=((xd+4)*(2*(ny+4)+j)+i)*5+l;//k=2;q=5
	t_phi[index_l_t]=phi[index_l];
	}
}

__global__ void boundary_xd( double *phi,double *t_phi )
{
	int j= blockIdx.x+1;
	int k=threadIdx.x+1;
	int i,index_l,index_l_t;
	int zd=nz/kp;
	int l_right[5]={1,7, 9,11,14};
	int l_left[5] ={2,8,10,12,13};

	for(int l=0;l<5;l++){
	i=2;
	index_l  =index_4d(i,j,k,l_left[l]);
	index_l_t=((ny+4)*(1*(zd+4)+k)+j)*5+l;//k=1;q=5
	t_phi[index_l_t]=phi[index_l];
	
	i=nx/ip+1;
	index_l  =index_4d(i,j,k,l_right[l]);
	index_l_t=((ny+4)*(2*(zd+4)+k)+j)*5+l;//k=2;q=5
	t_phi[index_l_t]=phi[index_l];
	}
}

__global__ void boundary_zd_undo( double *phi,double *t_phi)
{
	int i= blockIdx.x+1;
	int j=threadIdx.x+1;
	int k,index_l,index_l_t;
	int xd=nx/ip;
	int l_top[5]={5,11,13,15,18};
	int l_bot[5]={6,12,14,16,17};
	for(int l=0;l<5;l++){
	k=1;
	index_l  =index_4d(i,j,k,l_top[l]);
	index_l_t=((xd+4)*(0*(ny+4)+j)+i)*5+l;
	phi[index_l]=t_phi[index_l_t];
	
	k=nz/kp+2;
	index_l  =index_4d(i,j,k,l_bot[l]);
	index_l_t=((xd+4)*(3*(ny+4)+j)+i)*5+l;
	phi[index_l]=t_phi[index_l_t];
	}
}

__global__ void boundary_xd_undo( double *phi,double *t_phi)
{
	int j= blockIdx.x+1;
	int k=threadIdx.x+1;
	int i,index_l,index_l_t;
	int zd=nz/kp;
	int l_right[5]={1,7, 9,11,14};
	int l_left[5] ={2,8,10,12,13};
	for(int l=0;l<5;l++){
	i=1;
	index_l  =index_4d(i,j,k,l_right[l]);
	index_l_t=((ny+4)*(0*(zd+4)+k)+j)*5+l;
	phi[index_l]=t_phi[index_l_t];
	
	i=nx/ip+2;
	index_l  =index_4d(i,j,k,l_left[l]);
	index_l_t=((ny+4)*(3*(zd+4)+k)+j)*5+l;
	phi[index_l]=t_phi[index_l_t];
	}
}

__global__ void boundary_yd_in( double *g,double *h)
{
	int i= blockIdx.x+3;
	int k=threadIdx.x+3;
	int j,index_l;
	int distance=(ny)*(nx/ip+4);
	for(int l=0;l<q;l++){
	j=1;
	index_l=index_4d(i,j,k,l);
	g[index_l]=g[index_l+distance];
	h[index_l]=h[index_l+distance];
	j=ny+2;
	index_l=index_4d(i,j,k,l);
	g[index_l]=g[index_l-distance];
	h[index_l]=h[index_l-distance];
	}
}

__global__ void boundary_ym_bc( double *phi)
{
	int i =threadIdx.x+2;
	int zd=nz/kp;
	int kk[4]= {2,3,zd,zd+1};
	int distance=(ny)*(nx/ip+4);
	for (int t=0;t<4;t++){
	int k=kk[t];
	for (int j=0;j<2;j++){
	int index=index_3d(i,j,k);
	phi[index]=phi[index+distance];
	}
	for (int j=ny+2;j<ny+4;j++){
	int index=index_3d(i,j,k);
	phi[index]=phi[index-distance];
	}}
}

__global__ void boundary_ym_bc_x( double *phi)
{
	int k =threadIdx.x+2;
	int xd=nx/ip;
	int ii[4]= {2,3,xd,xd+1};
	int distance=(ny)*(xd+4);
	for (int t=0;t<4;t++){
	int i=ii[t];
	for (int j=0;j<2;j++){
	int index=index_3d(i,j,k);
	phi[index]=phi[index+distance];
	}
	for (int j=ny+2;j<ny+4;j++){
	int index=index_3d(i,j,k);
	phi[index]=phi[index-distance];
	}}
}

__global__ void boundary_ym_in( double *phi)
{
	int i= blockIdx.x+4;
	int k=threadIdx.x+4;
	int distance=(ny)*(nx/ip+4);

	for(int j=0;j<2;j++){
	int index=index_3d(i,j,k);
	phi[index]=phi[index+distance];
	}
	for(int j=ny+2;j<ny+4;j++){
	int index=index_3d(i,j,k);
	phi[index]=phi[index-distance];
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                 gradient                                                       //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void gradient_cen (	double *gra_phi, double *phi)
{
	const int i=threadIdx.x+2;
	const int j= blockIdx.x%ny+2;
	const int k= blockIdx.x/ny+2;
	const int index=index_3d(i,j,k);
	const double cs2_inv=3.0;
	
	double temp  =0.0;
	double temp_x=0.0;
	double temp_y=0.0;
	double temp_z=0.0;

	for(int l=1;l<q;l=l+2){
	double ex=eex[l];
	double ey=eey[l];
	double ez=eez[l];
	double wt=wwt[l];
	int	   et=eet[l];
	temp=2.0*wt*(phi[index+et]-phi[index-et]);
	temp_x=ex*temp+temp_x;
	temp_y=ey*temp+temp_y;
	temp_z=ez*temp+temp_z;
	}
	
	gra_phi[index_4d(i,j,k,0)]=temp_x*0.5*cs2_inv;
	gra_phi[index_4d(i,j,k,1)]=temp_y*0.5*cs2_inv;
	gra_phi[index_4d(i,j,k,2)]=temp_z*0.5*cs2_inv;
}

__device__ double grad_phie_c(double *phi,int index,int et)
{
	double ans;
	ans=(phi[index+et]-phi[index-et])*0.5;
	return ans;
}

__device__ double grad_phie_m(double *phi,int index,int et)
{
	double ans;
	ans=(-phi[index+2*et]+5.0*phi[index+et]-3.0*phi[index]-phi[index-et])*0.25;
	return ans;
}

__device__ double gradient_cen_x (	double *phi, int index )
{
	double ans=0.0;
	double cs2_inv=3.0;

#pragma unroll 9
	for(int l=1;l<q;l=l+2){
	double ex=eex[l];
	double wt=wwt[l];
	int	   et=eet[l];
	ans=ex*2.0*wt*(phi[index+et]-phi[index-et])+ans;
	}
	ans=ans*0.5*cs2_inv;
	return ans;
}

__device__ double gradient_cen_y (	double *phi, int index )
{
	double ans=0.0;
	double cs2_inv=3.0;

#pragma unroll 9
	for(int l=1;l<q;l=l+2){
	double ey=eey[l];
	double wt=wwt[l];
	int	   et=eet[l];
	ans=ey*2.0*wt*(phi[index+et]-phi[index-et])+ans;
	}
	ans=ans*0.5*cs2_inv;
	return ans;
}

__device__ double gradient_cen_z (	double *phi, int index )
{
	double ans=0.0;
	double cs2_inv=3.0;

#pragma unroll 9
	for(int l=1;l<q;l=l+2){
	double ez=eez[l];
	double wt=wwt[l];
	int	   et=eet[l];
	ans=ez*2.0*wt*(phi[index+et]-phi[index-et])+ans;
	}
	ans=ans*0.5*cs2_inv;
	return ans;
}

__device__ double gradient_mix_x ( double *phi, int index )
{
	double ans=0.0;
	double cs2_inv=3.0;

#pragma unroll 9
	for(int l=1;l<q;l=l+2){
	double ex=eex[l];
	double wt=wwt[l];
	int	   et=eet[l];
	ans=ex*wt*(-phi[index+2*et]+6.0*phi[index+et]-6.0*phi[index-et]+phi[index-2*et])+ans;
	}
	ans=ans*0.25*cs2_inv;
	return ans;
}

__device__ double gradient_mix_y ( double *phi, int index )
{
	double ans=0.0;
	double cs2_inv=3.0;

#pragma unroll 9
	for(int l=1;l<q;l=l+2){
	double ey=eey[l];
	double wt=wwt[l];
	int	   et=eet[l];
	ans=ey*wt*(-phi[index+2*et]+6.0*phi[index+et]-6.0*phi[index-et]+phi[index-2*et])+ans;
	}
	ans=ans*0.25*cs2_inv;
	return ans;
}

__device__ double gradient_mix_z ( double *phi, int index )
{
	double ans=0.0;
	double cs2_inv=3.0;

#pragma unroll 9
	for(int l=1;l<q;l=l+2){
	double ez=eez[l];
	double wt=wwt[l];
	int	   et=eet[l];
	ans=ez*wt*(-phi[index+2*et]+6.0*phi[index+et]-6.0*phi[index-et]+phi[index-2*et])+ans;
	}
	ans=ans*0.25*cs2_inv;
	return ans;
}

__device__ double laplace_phi (double *phi,int index)
{
	double ans=0.0;
	double phi_index=phi[index];
	double cs2_inv  =3.0;
	double dt_inv=1./dt;
	for(int l=1;l<q;l=l+2)
	{
	double wt=wwt[l];
	int	   et=eet[l];
	ans=2.0*wt*(phi[index+et]-2.0*phi_index+phi[index-et])+ans;
	}
	ans=ans*cs2_inv*dt_inv;
	return ans;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                chemical mu                                                     //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void	chemical(double *c,double *m,double kappa,double beta )
{
	int i=  threadIdx.x+2;
	int j=blockIdx.x%ny+2;
	int k=blockIdx.x/ny+2;
	int index=index_3d(i,j,k);
	double cl=c[index];
	m[index]=beta*(4.0*cl*cl*cl-6.0*cl*cl+2.0*cl)-kappa*laplace_phi( c,index );
}

__global__ void  chemical_bc( double *c,double *m,double kappa,double beta )
{
	int i=threadIdx.x+2;
	int j=blockIdx.x +2;
	int zd=nz/kp;
	int kk[4]= {2,3,zd,zd+1};
	for (int t=0;t<4;t++){
	int k=kk[t];
	int index=index_3d(i,j,k);
	double cl=c[index];
	m[index]=beta*(4.0*cl*cl*cl-6.0*cl*cl+2.0*cl)-kappa*laplace_phi( c,index );
	}
}

__global__ void  chemical_bc_x( double *c,double *m,double kappa,double beta )
{
	int k=threadIdx.x+4;
	int j=blockIdx.x +2;
	int xd=nx/ip;
	int ii[4]= {2,3,xd,xd+1};
	for (int t=0;t<4;t++){
	int i=ii[t];
	int index=index_3d(i,j,k);
	double cl=c[index];
	m[index]=beta*(4.0*cl*cl*cl-6.0*cl*cl+2.0*cl)-kappa*laplace_phi( c,index );
	}
}

__global__ void  chemical_in( double *c,double *m,double kappa,double beta )
{
	int i=threadIdx.x+4;
	int j=blockIdx.x%ny+2;
	int k=blockIdx.x/ny+4;
	int index=index_3d(i,j,k);
	double cl=c[index];

	m[index]=beta*(4.0*cl*cl*cl-6.0*cl*cl+2.0*cl)-kappa*laplace_phi( c,index );
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                 eq collision                                                   //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void	eq_collision(double *g,double *h,double *c,double *m,double *p,double gravity,double *gra_c,
							 double *gra_m,double *u,double *v,double *w,double mobi)
							 
{
	int i=threadIdx.x+2;
	int j= blockIdx.x%ny+2;
	int k= blockIdx.x/ny+2;
	int index=index_3d(i,j,k);
	
	double cs2_inv  =3.0;
	const double cs2=1.0/cs2_inv;
	
	double uu=u[index];
	double vv=v[index];
	double ww=w[index];
	double cc=c[index];
	double rr=cc*rho_l+(1.0-cc)*rho_g;
//	double tt=cc*tau_l+(1.0-cc)*tau_g;
	const double rr_inv=1.0/rr;
	double pp=p[index];
	double dr = rho_l-rho_g;
	
	double gr_cx_c=gra_c[index_4d(i,j,k,0)];
	double gr_cy_c=gra_c[index_4d(i,j,k,1)];
	double gr_cz_c=gra_c[index_4d(i,j,k,2)];
	double gr_mx_c=gra_m[index_4d(i,j,k,0)];
	double gr_my_c=gra_m[index_4d(i,j,k,1)];
	double gr_mz_c=gra_m[index_4d(i,j,k,2)];
	double gr_px_c=gradient_cen_x ( p,index );
	double gr_py_c=gradient_cen_y ( p,index );
	double gr_pz_c=gradient_cen_z ( p,index );
	
	double gr_cx_m=gradient_mix_x ( c,index );
	double gr_cy_m=gradient_mix_y ( c,index );
	double gr_cz_m=gradient_mix_z ( c,index );
	double gr_mx_m=gradient_mix_x ( m,index );
	double gr_my_m=gradient_mix_y ( m,index );
	double gr_mz_m=gradient_mix_z ( m,index );
	double gr_px_m=gradient_mix_x ( p,index );
	double gr_py_m=gradient_mix_y ( p,index );
	double gr_pz_m=gradient_mix_z ( p,index );
	
	double lap_mu   =laplace_phi( m,index );
	double udotu=uu*uu+vv*vv+ww*ww;
	
	for(int l=0;l<q;l++)
	{
	int index_l=index_4d(i,j,k,l);
	double ex=eex[l];
	double ey=eey[l];
	double ez=eez[l];
	double wt=wwt[l];
	int	   et=eet[l];
	
	double edotu=ex*uu+ey*vv+ez*ww;
	double uugly=edotu*cs2_inv+edotu*edotu*0.5*cs2_inv*cs2_inv-udotu*0.5*cs2_inv;
	double gamma=wt*(1.0+uugly);
	
	double u_et=u[index+et];
	double v_et=v[index+et];
	double w_et=w[index+et];
	double lap_mu_et=laplace_phi( m,index+et );
	double udotu_et=u_et*u_et+v_et*v_et+w_et*w_et;
	
	double edotu_et=ex*u_et+ey*v_et+ez*w_et;
	double uugly_et=edotu_et*cs2_inv+edotu_et*edotu_et*0.5*cs2_inv*cs2_inv-udotu_et*0.5*cs2_inv;
	double gamma_et=wt*(1.0+uugly_et);
	///////////////////////////////////////////////////////
	double geq_t=wt*(pp+rr*cs2*uugly);//geq
	double heq_t=cc*gamma;//heq
	///////////////////////////////////////////////////////
	double temp_cc = grad_phie_c( c,index,et ) - ( uu * gr_cx_c + vv * gr_cy_c + ww * gr_cz_c );
	double temp_mc = grad_phie_c( m,index,et ) - ( uu * gr_mx_c + vv * gr_my_c + ww * gr_mz_c );
	double temp_pc = grad_phie_c( p,index,et ) - ( uu * gr_px_c + vv * gr_py_c + ww * gr_pz_c );
	
	double temp_cm = grad_phie_m( c,index,et ) - ( uu * gr_cx_m + vv * gr_cy_m + ww * gr_cz_m );
	double temp_mm = grad_phie_m( m,index,et ) - ( uu * gr_mx_m + vv * gr_my_m + ww * gr_mz_m );
	double temp_pm = grad_phie_m( p,index,et ) - ( uu * gr_px_m + vv * gr_py_m + ww * gr_pz_m );
	
	double temp_z = ez*gravity-ww*gravity;
	///////////////////////////////////////////////////////
	double temp_gc = cs2*wt*uugly*temp_cc*dr-(cc*temp_mc+rr*temp_z)*gamma+ez*rho_l*gravity*wt;
	double temp_hc = temp_cc-cc*rr_inv*cs2_inv*(temp_pc+cc*temp_mc+(rr-rho_l)*temp_z);
	
	geq_t=geq_t-0.5*temp_gc;//geq_bar
	heq_t=heq_t-0.5*gamma*temp_hc;//heq_bar
	///////////////////////////////////////////////////////
	double temp_gm = cs2*wt*uugly*temp_cm*dr-(cc*temp_mm+rr*temp_z)*gamma+ez*rho_l*gravity*wt;
	double temp_hm = temp_cm-cc*rr_inv*cs2_inv*(temp_pm+cc*temp_mm+(rr-rho_l)*temp_z);
	temp_hm = 0.5*dt*mobi*( gamma*lap_mu + gamma_et*lap_mu_et )+temp_hm*gamma;
	////////////////////////collision//////////////////////////////
	g[index_l] = geq_t+temp_gm;
	h[index_l] = heq_t+temp_hm;
	}
}

__global__ void	eq_collision_bc(double *g,double *h,double *c,double *m,double *p,double gravity,double *gra_c,
							    double *gra_m,double *u,double *v,double *w,double mobi)
{
	int i=threadIdx.x+2;
	int j= blockIdx.x+2;
	int zd=nz/kp;
	double cs2_inv  =3.0;
	double cs2=1.0/cs2_inv;
	double dr = rho_l-rho_g;
	
	for(int k=2;k<zd+2;k=k+zd-1)
	{
	int index=index_3d(i,j,k);
	double uu=u[index];
	double vv=v[index];
	double ww=w[index];
	double cc=c[index];
	double ceq=cc;
  	if     (cc < 0)ceq=0;
	else if(cc > 1)ceq=1;
	else           ceq=cc;
	double rr=cc*rho_l+(1.0-cc)*rho_g;
	double tt=cc*tau_l+(1.0-cc)*tau_g;
	double rr_inv=1.0/rr;
	double pp=p[index];
	
	double gr_cx_c=gra_c[index_4d(i,j,k,0)];
	double gr_cy_c=gra_c[index_4d(i,j,k,1)];
	double gr_cz_c=gra_c[index_4d(i,j,k,2)];
	double gr_mx_c=gra_m[index_4d(i,j,k,0)];
	double gr_my_c=gra_m[index_4d(i,j,k,1)];
	double gr_mz_c=gra_m[index_4d(i,j,k,2)];
	double gr_px_c=gradient_cen_x ( p,index );
	double gr_py_c=gradient_cen_y ( p,index );
	double gr_pz_c=gradient_cen_z ( p,index );
	
	double gr_cx_m=gradient_mix_x ( c,index );
	double gr_cy_m=gradient_mix_y ( c,index );
	double gr_cz_m=gradient_mix_z ( c,index );
	double gr_mx_m=gradient_mix_x ( m,index );
	double gr_my_m=gradient_mix_y ( m,index );
	double gr_mz_m=gradient_mix_z ( m,index );
	double gr_px_m=gradient_mix_x ( p,index );
	double gr_py_m=gradient_mix_y ( p,index );
	double gr_pz_m=gradient_mix_z ( p,index );
	
	double lap_mu =laplace_phi( m,index );
	double udotu=uu*uu+vv*vv+ww*ww;
	
	for(int l=0;l<q;l++)
	{
	int index_l=index_4d(i,j,k,l);
	double ex=eex[l];
	double ey=eey[l];
	double ez=eez[l];
	double wt=wwt[l];
	int	   et=eet[l];
	
	double edotu=ex*uu+ey*vv+ez*ww;
	double uugly=edotu*cs2_inv+edotu*edotu*0.5*cs2_inv*cs2_inv-udotu*0.5*cs2_inv;
	double gamma=wt*(1.0+uugly);
	
	double u_et=u[index+et];
	double v_et=v[index+et];
	double w_et=w[index+et];
	double lap_mu_et=laplace_phi( m,index+et );
	double udotu_et=u_et*u_et+v_et*v_et+w_et*w_et;
	
	double edotu_et=ex*u_et+ey*v_et+ez*w_et;
	double uugly_et=edotu_et*cs2_inv+edotu_et*edotu_et*0.5*cs2_inv*cs2_inv-udotu_et*0.5*cs2_inv;
	double gamma_et=wt*(1.0+uugly_et);
	///////////////////////////////////////////////////////
	double geq_t=wt*(pp+rr*cs2*uugly);//geq
	double heq_t=ceq*gamma;//heq
	///////////////////////////////////////////////////////
	double temp_cc = grad_phie_c( c,index,et ) - ( uu * gr_cx_c + vv * gr_cy_c + ww * gr_cz_c );
	double temp_mc = grad_phie_c( m,index,et ) - ( uu * gr_mx_c + vv * gr_my_c + ww * gr_mz_c );
	double temp_pc = grad_phie_c( p,index,et ) - ( uu * gr_px_c + vv * gr_py_c + ww * gr_pz_c );
	
	double temp_cm = grad_phie_m( c,index,et ) - ( uu * gr_cx_m + vv * gr_cy_m + ww * gr_cz_m );
	double temp_mm = grad_phie_m( m,index,et ) - ( uu * gr_mx_m + vv * gr_my_m + ww * gr_mz_m );
	double temp_pm = grad_phie_m( p,index,et ) - ( uu * gr_px_m + vv * gr_py_m + ww * gr_pz_m );
	
	double temp_z = ez*gravity-ww*gravity;
	///////////////////////////////////////////////////////
	double temp_gc = cs2*wt*uugly*temp_cc*dr-(cc*temp_mc+rr*temp_z)*gamma+ez*rho_l*gravity*wt;
	double temp_hc = temp_cc-cc*rr_inv*cs2_inv*(temp_pc+cc*temp_mc+rr*temp_z);
	
	geq_t=geq_t-0.5*temp_gc;//geq_bar
	heq_t=heq_t-0.5*temp_hc*gamma;//heq_bar
	///////////////////////////////////////////////////////
	double temp_gm = cs2*wt*uugly*temp_cm*dr-(cc*temp_mm+rr*temp_z)*gamma+ez*rho_l*gravity*wt;
	double temp_hm = temp_cm-cc*rr_inv*cs2_inv*(temp_pm+cc*temp_mm+rr*temp_z);
	temp_hm = 0.5*dt*mobi*( gamma*lap_mu + gamma_et*lap_mu_et )+temp_hm*gamma;
	////////////////////////collision//////////////////////////////
	g[index_l] = g[index_l]*(1.0-1.0/(tt    +0.5))+geq_t/(tt    +0.5)+temp_gm;
	h[index_l] = h[index_l]*(1.0-1.0/(tau_h +0.5))+heq_t/(tau_h +0.5)+temp_hm;
	}
	}
}

__global__ void	eq_collision_bc_x(double *g,double *h,double *c,double *m,double *p,double gravity,double *gra_c,
							      double *gra_m,double *u,double *v,double *w,double mobi)
{
	int k=threadIdx.x+3;
	int j= blockIdx.x+2;
	int xd=nx/ip;
	double cs2_inv  =3.0;
	double cs2      =1.0/cs2_inv;
	double dr = rho_l-rho_g;
	
	for(int i=2;i<xd+2;i=i+xd-1)
	{
	int index=index_3d(i,j,k);
	double uu=u[index];
	double vv=v[index];
	double ww=w[index];
	double cc=c[index];
	double ceq=cc;
  	if     (cc < 0)ceq=0;
	else if(cc > 1)ceq=1;
	else           ceq=cc;
	double rr=cc*rho_l+(1.0-cc)*rho_g;
	double tt=cc*tau_l+(1.0-cc)*tau_g;
	double rr_inv=1.0/rr;
	double pp=p[index];
	
	double gr_cx_c=gra_c[index_4d(i,j,k,0)];
	double gr_cy_c=gra_c[index_4d(i,j,k,1)];
	double gr_cz_c=gra_c[index_4d(i,j,k,2)];
	double gr_mx_c=gra_m[index_4d(i,j,k,0)];
	double gr_my_c=gra_m[index_4d(i,j,k,1)];
	double gr_mz_c=gra_m[index_4d(i,j,k,2)];
	double gr_px_c=gradient_cen_x ( p,index );
	double gr_py_c=gradient_cen_y ( p,index );
	double gr_pz_c=gradient_cen_z ( p,index );
	
	double gr_cx_m=gradient_mix_x ( c,index );
	double gr_cy_m=gradient_mix_y ( c,index );
	double gr_cz_m=gradient_mix_z ( c,index );
	double gr_mx_m=gradient_mix_x ( m,index );
	double gr_my_m=gradient_mix_y ( m,index );
	double gr_mz_m=gradient_mix_z ( m,index );
	double gr_px_m=gradient_mix_x ( p,index );
	double gr_py_m=gradient_mix_y ( p,index );
	double gr_pz_m=gradient_mix_z ( p,index );
	
	double lap_mu =laplace_phi( m,index );
	double udotu=uu*uu+vv*vv+ww*ww;
	
	for(int l=0;l<q;l++)
	{
	int index_l=index_4d(i,j,k,l);
	double ex=eex[l];
	double ey=eey[l];
	double ez=eez[l];
	double wt=wwt[l];
	int	   et=eet[l];
	
	double edotu=ex*uu+ey*vv+ez*ww;
	double uugly=edotu*cs2_inv+edotu*edotu*0.5*cs2_inv*cs2_inv-udotu*0.5*cs2_inv;
	double gamma=wt*(1.0+uugly);
	
	double u_et=u[index+et];
	double v_et=v[index+et];
	double w_et=w[index+et];
	double lap_mu_et=laplace_phi( m,index+et );
	double udotu_et=u_et*u_et+v_et*v_et+w_et*w_et;
	
	double edotu_et=ex*u_et+ey*v_et+ez*w_et;
	double uugly_et=edotu_et*cs2_inv+edotu_et*edotu_et*0.5*cs2_inv*cs2_inv-udotu_et*0.5*cs2_inv;
	double gamma_et=wt*(1.0+uugly_et);
	///////////////////////////////////////////////////////
	double geq_t=wt*(pp+rr*cs2*uugly);//geq
	double heq_t=ceq*gamma;//heq
	///////////////////////////////////////////////////////
	double temp_cc = grad_phie_c( c,index,et ) - ( uu * gr_cx_c + vv * gr_cy_c + ww * gr_cz_c );
	double temp_mc = grad_phie_c( m,index,et ) - ( uu * gr_mx_c + vv * gr_my_c + ww * gr_mz_c );
	double temp_pc = grad_phie_c( p,index,et ) - ( uu * gr_px_c + vv * gr_py_c + ww * gr_pz_c );
	
	double temp_cm = grad_phie_m( c,index,et ) - ( uu * gr_cx_m + vv * gr_cy_m + ww * gr_cz_m );
	double temp_mm = grad_phie_m( m,index,et ) - ( uu * gr_mx_m + vv * gr_my_m + ww * gr_mz_m );
	double temp_pm = grad_phie_m( p,index,et ) - ( uu * gr_px_m + vv * gr_py_m + ww * gr_pz_m );
	
	double temp_z = ez*gravity-ww*gravity;
	///////////////////////////////////////////////////////
	double temp_gc = cs2*wt*uugly*temp_cc*dr-(cc*temp_mc+rr*temp_z)*gamma+ez*rho_l*gravity*wt;
	double temp_hc = temp_cc-cc*rr_inv*cs2_inv*(temp_pc+cc*temp_mc+rr*temp_z);
	
	geq_t=geq_t-0.5*temp_gc;//geq_bar
	heq_t=heq_t-0.5*temp_hc*gamma;//heq_bar
	///////////////////////////////////////////////////////
	double temp_gm = cs2*wt*uugly*temp_cm*dr-(cc*temp_mm+rr*temp_z)*gamma+ez*rho_l*gravity*wt;
	double temp_hm = temp_cm-cc*rr_inv*cs2_inv*(temp_pm+cc*temp_mm+rr*temp_z);
	temp_hm = 0.5*dt*mobi*( gamma*lap_mu + gamma_et*lap_mu_et )+temp_hm*gamma;
	////////////////////////collision//////////////////////////////
	g[index_l] = g[index_l]*(1.0-1.0/(tt    +0.5))+geq_t/(tt    +0.5)+temp_gm;
	h[index_l] = h[index_l]*(1.0-1.0/(tau_h +0.5))+heq_t/(tau_h +0.5)+temp_hm;
	}
	}
}

__global__ void	eq_collision_in(double *g,double *h,double *c,double *m,double *p,double gravity,double *gra_c,
							    double *gra_m,double *u,double *v,double *w,double mobi)
{
	int i=threadIdx.x+3;
	int j= blockIdx.x%ny+2;
	int k= blockIdx.x/ny+3;
	int index=index_3d(i,j,k);
	
	double cs2_inv  =3.0;
	double cs2=1.0/cs2_inv;
	double uu=u[index];
	double vv=v[index];
	double ww=w[index];
	double cc=c[index];
	double ceq=cc;
  	if     (cc < 0)ceq=0;
	else if(cc > 1)ceq=1;
	else           ceq=cc;
	double rr=cc*rho_l+(1.0-cc)*rho_g;
	double tt=cc*tau_l+(1.0-cc)*tau_g;
	double rr_inv=1.0/rr;
	double pp=p[index];
	double dr = rho_l-rho_g;
	
	double gr_cx_c=gra_c[index_4d(i,j,k,0)];
	double gr_cy_c=gra_c[index_4d(i,j,k,1)];
	double gr_cz_c=gra_c[index_4d(i,j,k,2)];
	double gr_mx_c=gra_m[index_4d(i,j,k,0)];
	double gr_my_c=gra_m[index_4d(i,j,k,1)];
	double gr_mz_c=gra_m[index_4d(i,j,k,2)];
	double gr_px_c=gradient_cen_x ( p,index );
	double gr_py_c=gradient_cen_y ( p,index );
	double gr_pz_c=gradient_cen_z ( p,index );
	
	double gr_cx_m=gradient_mix_x ( c,index );
	double gr_cy_m=gradient_mix_y ( c,index );
	double gr_cz_m=gradient_mix_z ( c,index );
	double gr_mx_m=gradient_mix_x ( m,index );
	double gr_my_m=gradient_mix_y ( m,index );
	double gr_mz_m=gradient_mix_z ( m,index );
	double gr_px_m=gradient_mix_x ( p,index );
	double gr_py_m=gradient_mix_y ( p,index );
	double gr_pz_m=gradient_mix_z ( p,index );
	
	double lap_mu =laplace_phi( m,index );
	double udotu=uu*uu+vv*vv+ww*ww;
	
	for(int l=0;l<q;l++)
	{
	int index_l=index_4d(i,j,k,l);
	double ex=eex[l];
	double ey=eey[l];
	double ez=eez[l];
	double wt=wwt[l];
	int	   et=eet[l];
	
	double edotu=ex*uu+ey*vv+ez*ww;
	double uugly=edotu*cs2_inv+edotu*edotu*0.5*cs2_inv*cs2_inv-udotu*0.5*cs2_inv;
	double gamma=wt*(1.0+uugly);
	
	double u_et=u[index+et];
	double v_et=v[index+et];
	double w_et=w[index+et];
	double lap_mu_et=laplace_phi( m,index+et );
	double udotu_et=u_et*u_et+v_et*v_et+w_et*w_et;
	
	double edotu_et=ex*u_et+ey*v_et+ez*w_et;
	double uugly_et=edotu_et*cs2_inv+edotu_et*edotu_et*0.5*cs2_inv*cs2_inv-udotu_et*0.5*cs2_inv;
	double gamma_et=wt*(1.0+uugly_et);
	///////////////////////////////////////////////////////
	double geq_t=wt*(pp+rr*cs2*uugly);//geq
	double heq_t=ceq*gamma;//heq
	///////////////////////////////////////////////////////
	double temp_cc = grad_phie_c( c,index,et ) - ( uu * gr_cx_c + vv * gr_cy_c + ww * gr_cz_c );
	double temp_mc = grad_phie_c( m,index,et ) - ( uu * gr_mx_c + vv * gr_my_c + ww * gr_mz_c );
	double temp_pc = grad_phie_c( p,index,et ) - ( uu * gr_px_c + vv * gr_py_c + ww * gr_pz_c );
	
	double temp_cm = grad_phie_m( c,index,et ) - ( uu * gr_cx_m + vv * gr_cy_m + ww * gr_cz_m );
	double temp_mm = grad_phie_m( m,index,et ) - ( uu * gr_mx_m + vv * gr_my_m + ww * gr_mz_m );
	double temp_pm = grad_phie_m( p,index,et ) - ( uu * gr_px_m + vv * gr_py_m + ww * gr_pz_m );
	
	double temp_z = ez*gravity-ww*gravity;
	///////////////////////////////////////////////////////
	double temp_gc = cs2*wt*uugly*temp_cc*dr-(cc*temp_mc+rr*temp_z)*gamma+ez*rho_l*gravity*wt;
	double temp_hc = temp_cc-cc*rr_inv*cs2_inv*(temp_pc+cc*temp_mc+rr*temp_z);
	
	geq_t=geq_t-0.5*temp_gc;//geq_bar
	heq_t=heq_t-0.5*temp_hc*gamma;//heq_bar
	///////////////////////////////////////////////////////
	double temp_gm = cs2*wt*uugly*temp_cm*dr-(cc*temp_mm+rr*temp_z)*gamma+ez*rho_l*gravity*wt;
	double temp_hm = temp_cm-cc*rr_inv*cs2_inv*(temp_pm+cc*temp_mm+rr*temp_z);
	temp_hm = 0.5*dt*mobi*( gamma*lap_mu + gamma_et*lap_mu_et )+temp_hm*gamma;
	////////////////////////collision//////////////////////////////
	g[index_l] = g[index_l]*(1.0-1.0/(tt    +0.5))+geq_t/(tt    +0.5)+temp_gm;
	h[index_l] = h[index_l]*(1.0-1.0/(tau_h +0.5))+heq_t/(tau_h +0.5)+temp_hm;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                    macro                                                       //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void macro_h_bc(double *h,double *h_next,double *c)
{
	int i=threadIdx.x+2;
	int j= blockIdx.x+2;
	int zd=nz/kp;
	int kk[4]= {2,3,zd,zd+1};
	for (int t=0;t<4;t++){
	int k=kk[t];
	int index=index_3d(i,j,k);
	double sum_c=0.0;
	for(int l=0;l<q;l++){
	int index_l =index_4d(i,j,k,l);
	int et=eet[l];
	sum_c=h[index_l-et]+sum_c;
	h_next[index_l]=h[index_l-et];
	}
	c[index]=sum_c;
	}
}

__global__ void macro_h_bc_x(double *h,double *h_next,double *c)
{
	int k=threadIdx.x+4;
	int j= blockIdx.x+2;
	int xd=nx/ip;
	int ii[4]= {2,3,xd,xd+1};
	for (int t=0;t<4;t++){
	int i=ii[t];
	int index=index_3d(i,j,k);
	double sum_c=0.0;
	for(int l=0;l<q;l++){
	int index_l =index_4d(i,j,k,l);
	int et=eet[l];
	sum_c=h[index_l-et]+sum_c;
	h_next[index_l]=h[index_l-et];
	}
	c[index]=sum_c;
	}
}

__global__ void macro_h_in(double *h,double *h_next,double *c)
{
	int i=threadIdx.x+4;
	int j= blockIdx.x%ny+2;
	int k= blockIdx.x/ny+4;
	int index=index_3d(i,j,k);
	double sum_c=0.0;

	for(int l=0;l<q;l++){
	int index_l =index_4d(i,j,k,l);
	int et=eet[l];
	sum_c=h[index_l-et]+sum_c;
	h_next[index_l]=h[index_l-et];
	}
	c[index]=sum_c;
}

__global__ void	macro_g_bc(double *g,double *g_next,double *c,double *m,double *p,double *gra_c,double *gra_m,double *u,double *v,double *w)
{
	int i=threadIdx.x+2;
	int j= blockIdx.x+2;
	int zd=nz/kp;
	int kk[4]= {2,3,zd,zd+1};
	double dr=rho_l-rho_g;
	double cs2_inv=3.0;
	double cs2=1.0/cs2_inv;
	for (int t=0;t<4;t++){
	int k=kk[t];
	int index=index_3d(i,j,k);
	double cc=c[index];
	double rr=cc*rho_l+((double)1.0-cc)*rho_g;
 	double gr_rx_c=gra_c[index_4d(i,j,k,0)]*dr;
	double gr_ry_c=gra_c[index_4d(i,j,k,1)]*dr;
	double gr_rz_c=gra_c[index_4d(i,j,k,2)]*dr;
	double gr_mx_c=gra_m[index_4d(i,j,k,0)];
	double gr_my_c=gra_m[index_4d(i,j,k,1)];
	double gr_mz_c=gra_m[index_4d(i,j,k,2)];
	
	double sum_u=0.0;
	double sum_v=0.0;
	double sum_w=0.0;
	double sum_p=0.0;
	
	for(int l=0;l<q;l++)
	{
	int index_l=index_4d(i,j,k,l);
	double ex=eex[l];
	double ey=eey[l];
	double ez=eez[l];
	int	   et=eet[l];
	
	double temp_g=g[index_l-et];
	sum_u=ex*temp_g+sum_u;
	sum_v=ey*temp_g+sum_v;
	sum_w=ez*temp_g+sum_w;
	sum_p=   temp_g+sum_p;
	g_next[index_l]=temp_g;
	}
	double uu=(sum_u*cs2_inv-0.5*dt*cc*gr_mx_c)/rr;
	double vv=(sum_v*cs2_inv-0.5*dt*cc*gr_my_c)/rr;
	double ww=(sum_w*cs2_inv-0.5*dt*cc*gr_mz_c)/rr;
	u[index]=uu;
	v[index]=vv;
	w[index]=ww;
	p[index]=sum_p+0.5*dt*(uu*gr_rx_c+vv*gr_ry_c+ww*gr_rz_c)*cs2;
	}
}

__global__ void	macro_g_bc_x(double *g,double *g_next,double *c,double *m,double *p,double *gra_c,double *gra_m,double *u,double *v,double *w)
{
	int k=threadIdx.x+4;
	int j= blockIdx.x+2;
	int xd=nx/ip;
	int ii[4]= {2,3,xd,xd+1};
	double cs2_inv=3.0;
	double cs2=1.0/cs2_inv;
	double dr=rho_l-rho_g;

	for (int t=0;t<4;t++){
	int i=ii[t];
	int index=index_3d(i,j,k);
	double cc=c[index];
	double rr=cc*rho_l+((double)1.0-cc)*rho_g;
 	double gr_rx_c=gra_c[index_4d(i,j,k,0)]*dr;
	double gr_ry_c=gra_c[index_4d(i,j,k,1)]*dr;
	double gr_rz_c=gra_c[index_4d(i,j,k,2)]*dr;
	double gr_mx_c=gra_m[index_4d(i,j,k,0)];
	double gr_my_c=gra_m[index_4d(i,j,k,1)];
	double gr_mz_c=gra_m[index_4d(i,j,k,2)];
	
	double sum_u=0.0;
	double sum_v=0.0;
	double sum_w=0.0;
	double sum_p=0.0;
	
	for(int l=0;l<q;l++)
	{
	int index_l=index_4d(i,j,k,l);
	double ex=eex[l];
	double ey=eey[l];
	double ez=eez[l];
	int	   et=eet[l];
	
	double temp_g=g[index_l-et];
	sum_u=ex*temp_g+sum_u;
	sum_v=ey*temp_g+sum_v;
	sum_w=ez*temp_g+sum_w;
	sum_p=   temp_g+sum_p;
	g_next[index_l]=temp_g;
	}
	double uu=(sum_u*cs2_inv-0.5*dt*cc*gr_mx_c)/rr;
	double vv=(sum_v*cs2_inv-0.5*dt*cc*gr_my_c)/rr;
	double ww=(sum_w*cs2_inv-0.5*dt*cc*gr_mz_c)/rr;
	u[index]=uu;
	v[index]=vv;
	w[index]=ww;
	p[index]=sum_p+0.5*dt*(uu*gr_rx_c+vv*gr_ry_c+ww*gr_rz_c)*cs2;
	}
}

__global__ void	macro_g_in( double *g, double *g_next,double *c,double *m,double *p,double *gra_c,double *gra_m,double *u,double *v,double *w)
{
	int i=threadIdx.x+4;
	int j= blockIdx.x%ny+2;
	int k= blockIdx.x/ny+4;
	int index=index_3d(i,j,k);
	double cs2_inv=3.0;
	double cs2=1.0/cs2_inv;
	double cc=c[index];
	double rr=cc*rho_l+((double)1.0-cc)*rho_g;
	double dr=rho_l-rho_g;
	
	double gr_rx_c=gra_c[index_4d(i,j,k,0)]*dr;
	double gr_ry_c=gra_c[index_4d(i,j,k,1)]*dr;
	double gr_rz_c=gra_c[index_4d(i,j,k,2)]*dr;
	double gr_mx_c=gra_m[index_4d(i,j,k,0)];
	double gr_my_c=gra_m[index_4d(i,j,k,1)];
	double gr_mz_c=gra_m[index_4d(i,j,k,2)];
	
	double sum_u=0.0;
	double sum_v=0.0;
	double sum_w=0.0;
	double sum_p=0.0;
	
	for(int l=0;l<q;l++){
	int index_l =index_4d(i,j,k,l);
	double ex=eex[l];
	double ey=eey[l];
	double ez=eez[l];
	int	   et=eet[l];
	
	double temp_g=g[index_l-et];
	sum_u=ex*temp_g+sum_u;
	sum_v=ey*temp_g+sum_v;
	sum_w=ez*temp_g+sum_w;
	sum_p=   temp_g+sum_p;
	g_next[index_l]=temp_g;
	}
	
	double uu=(sum_u*cs2_inv-0.5*dt*cc*gr_mx_c)/rr;
	double vv=(sum_v*cs2_inv-0.5*dt*cc*gr_my_c)/rr;
	double ww=(sum_w*cs2_inv-0.5*dt*cc*gr_mz_c)/rr;
	u[index]=uu;
	v[index]=vv;
	w[index]=ww;
	p[index]=sum_p+0.5*dt*(uu*gr_rx_c+vv*gr_ry_c+ww*gr_rz_c)*cs2;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                      post                                                      //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void	p_real(double *c,double *p,double *a,double beta,double kappa,double *gra_c)
{
	int i=threadIdx.x+2;
	int j= blockIdx.x%ny+2;
	int k= blockIdx.x/ny+2;
	int index=index_3d(i,j,k);
	
	double gr_cx_c=gra_c[index_4d(i,j,k,0)];
	double gr_cy_c=gra_c[index_4d(i,j,k,1)];
	double gr_cz_c=gra_c[index_4d(i,j,k,2)];
	double la_c   =laplace_phi(c,index );
	double cc=c[index];
	double pp=p[index];
	
	
	double th,cu,e0;
	e0=beta*cc*cc*(cc-1)*(cc-1);
	th=cc*beta*(4*cc*cc*cc-6*cc*cc+2*cc)-e0;
	cu=-kappa*cc*la_c+0.5*kappa*(gr_cx_c*gr_cx_c+gr_cy_c*gr_cy_c+gr_cz_c*gr_cz_c);
	a[index]=pp+th+cu;
}

double maxvalue(double *phi, int* indexx)
{
	double max=0.0;
	int i,j,k;
	for(i=0;i<nx;i++){
	for(j=0;j<ny;j++){
	for(k=0;k<nz;k++){
	int index=nx*(k*ny+j)+i;
	if(max < phi[index]){
		max=phi[index];
		*indexx=index;
	}}}}
	return max;
}

void max_w(double *c,double *w,double *max)
{
	*max=0.0;
	int i,j,k;
	for(i=0;i<nx;i++){
	for(j=0;j<ny;j++){
	for(k=0;k<nz;k++){
	int index=nx*(k*ny+j)+i;
	if(*max < w[index]){
	   *max = w[index];
	}}}}
}

double minvalue(double *phi, int* indexx)
{
	double min=100.0;
	int i,j,k;
	for(i=0;i<nx;i++){
	for(j=0;j<ny;j++){
	for(k=0;k<nz;k++){
	int index=nx*(k*ny+j)+i;
	if(min > phi[index]){
		min=phi[index];
		*indexx=index;
	}}}}
	return min;
}

void Reynolds_Time(double w, double *Re, int step)
{
	Re[step/2-1]=2*radd*3/tau_l*w;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                      main                                                      //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[])
{
	int i,j,k,index;
	//define matrix(會切割的)
	double *c_d_h,*c_f_h,*c_fdo_h,*c_d,*c; // dicom & final & transfered on host/ orifinal & transfered on device
	double *m_d_h,*m_f_h,*m_fdo_h,*m_d,*m;
	double *b_d_h,*b_f_h,*b_fdo_h,*b_d,*b; // wettability
	double *p_d_h,*p_f_h,*p_fdo_h,*p_d,*p;
	double *u_d_h,*u_f_h,*u_fdo_h,*u_d,*u;
	double *v_d_h,*v_f_h,*v_fdo_h,*v_d,*v;
	double *w_d_h,*w_f_h,*w_fdo_h,*w_d,*w;
	double *a_d_h,*a_f_h,*a_fdo_h,*a_d,*a; //total pressure
	
	double *xz_d_h,*xz_f_h,*xz_fdo_h,*xz_d;
	
	
	//define matrix(不會切割的)
	int    *et_h;//方向
	double *ex_h,*ey_h,*ez_h,*wt_h;
	double *h,*h_t;
	double *g,*g_t;
	//gradient matrix
	double *gra_c;
	double *gra_m;
	//define matrix(邊界交換的小矩陣)
	double *t_c_h,*t_c;
	double *t_m_h,*t_m;
	double *t_b_h,*t_b;
	double *t_p_h,*t_p;
	double *t_u_h,*t_u;
	double *t_v_h,*t_v;
	double *t_w_h,*t_w;
	double *t_g_h,*t_g;
	double *t_h_h,*t_h;
	
	double *t_c_x_h,*t_c_x;
	double *t_m_x_h,*t_m_x;
	double *t_b_x_h,*t_b_x;
	double *t_p_x_h,*t_p_x;
	double *t_u_x_h,*t_u_x;
	double *t_v_x_h,*t_v_x;
	double *t_w_x_h,*t_w_x;
	double *t_g_x_h,*t_g_x;
	double *t_h_x_h,*t_h_x;
	
	double *lx,*lz;
	double *Re;

////mpi
	int nproc,myid;
	int l_nbr, b_nbr, r_nbr, t_nbr, my_coord[ndim], iroot, itag;
	int ipart[ndim],periods[ndim],sideways,updown,right,up,reorder;
	int n_f;
	MPI_Status istat[8];
	MPI_Comm comm;
	
	MPI_Init( &argc, &argv );
	MPI_Comm_size(MPI_COMM_WORLD, &nproc);
	comm = MPI_COMM_WORLD;
	
	ipart[0]=ip;
	ipart[1]=kp;
	periods[0]=1;
	periods[1]=1;
	reorder=1;
	
	MPI_Cart_create(MPI_COMM_WORLD,ndim,ipart,periods,reorder,&comm);
	MPI_Comm_rank(comm,&myid);
	MPI_Cart_coords(comm,myid,ndim,my_coord);
	
	sideways=0;
	updown=1;
	right=1;
	up=1;
	
	MPI_Cart_shift(comm,sideways,right,&l_nbr,&r_nbr);
	MPI_Cart_shift(comm,updown  ,up   ,&b_nbr,&t_nbr);
	
	n_f=nx/ip*ny*nz/kp;
	if(myid==0){
	printf("===============================================================\n");
	printf("Checking devices...\n");
	}
	MPI_Barrier(MPI_COMM_WORLD);
	printf("NPROC,MYID,i,k=%d\t%d\t%d\t%d\t\n",nproc,myid,my_coord[0],my_coord[1]);
	MPI_Barrier(MPI_COMM_WORLD);
	
	hipSetDevice(myid%DPN);
	
////memory allocate on cpu
	
	int size_final = nx*ny*nz;
	int size_dicom = (nx/ip+4)*(ny+4)*(nz/kp+4);
	int size_difun = (nx/ip+4)*(ny+4)*(nz/kp+4)*q;
	int size_allgr = (nx/ip+4)*(ny+4)*(nz/kp+4)*3;//(x+y+z)
	
	int tran_mac_1 = (nx/ip+4)*(ny+4)*4*1; //u,v,w
	int tran_mac_2 = (nx/ip+4)*(ny+4)*4*2; //c,m,b,p
	int tran_difun = (nx/ip+4)*(ny+4)*4*5;//5個方向
	
	int tran_mac_1_x = (nz/kp+4)*(ny+4)*4*1; //u,v,w x face
	int tran_mac_2_x = (nz/kp+4)*(ny+4)*4*2; //c,m,b,p x face
	int tran_difun_x = (nz/kp+4)*(ny+4)*4*5;//5個方向 x face
	
	hipHostMalloc((void**)&c_d_h ,sizeof(double)*size_dicom);
	hipHostMalloc((void**)&m_d_h ,sizeof(double)*size_dicom);
	hipHostMalloc((void**)&b_d_h ,sizeof(double)*size_dicom);
	hipHostMalloc((void**)&p_d_h ,sizeof(double)*size_dicom);
	hipHostMalloc((void**)&u_d_h ,sizeof(double)*size_dicom);
	hipHostMalloc((void**)&v_d_h ,sizeof(double)*size_dicom);
	hipHostMalloc((void**)&w_d_h ,sizeof(double)*size_dicom); 
	hipHostMalloc((void**)&a_d_h ,sizeof(double)*size_dicom); 
	
	hipHostMalloc((void**)&et_h ,sizeof(double)* q ); 
	hipHostMalloc((void**)&ex_h ,sizeof(double)* q ); 
	hipHostMalloc((void**)&ey_h ,sizeof(double)* q ); 
	hipHostMalloc((void**)&ez_h ,sizeof(double)* q );
	hipHostMalloc((void**)&wt_h ,sizeof(double)* q );
	
	hipHostMalloc((void**)&t_c_h	,sizeof(double)* tran_mac_2 );
	hipHostMalloc((void**)&t_m_h	,sizeof(double)* tran_mac_2 );
	hipHostMalloc((void**)&t_b_h	,sizeof(double)* tran_mac_2 );
	hipHostMalloc((void**)&t_p_h	,sizeof(double)* tran_mac_2 );
	hipHostMalloc((void**)&t_u_h	,sizeof(double)* tran_mac_1 );
	hipHostMalloc((void**)&t_v_h	,sizeof(double)* tran_mac_1 );
	hipHostMalloc((void**)&t_w_h	,sizeof(double)* tran_mac_1 );
	hipHostMalloc((void**)&t_g_h	,sizeof(double)* tran_difun ); 
	hipHostMalloc((void**)&t_h_h	,sizeof(double)* tran_difun );
	
	hipHostMalloc((void**)&t_c_x_h	,sizeof(double)* tran_mac_2_x );
	hipHostMalloc((void**)&t_m_x_h	,sizeof(double)* tran_mac_2_x );
	hipHostMalloc((void**)&t_b_x_h	,sizeof(double)* tran_mac_2_x );
	hipHostMalloc((void**)&t_p_x_h	,sizeof(double)* tran_mac_2_x );
	hipHostMalloc((void**)&t_u_x_h	,sizeof(double)* tran_mac_1_x );
	hipHostMalloc((void**)&t_v_x_h	,sizeof(double)* tran_mac_1_x );
	hipHostMalloc((void**)&t_w_x_h	,sizeof(double)* tran_mac_1_x );
	hipHostMalloc((void**)&t_g_x_h	,sizeof(double)* tran_difun_x ); 
	hipHostMalloc((void**)&t_h_x_h	,sizeof(double)* tran_difun_x );	

	hipHostMalloc((void**)&xz_d_h  ,sizeof(double)*(nx/ip+4)*(nz/kp+4)); 
	
///////////////////////////////////////////////////////////////////////////////////////////
//                                         zz                                            //
///////////////////////////////////////////////////////////////////////////////////////////
	int step=0;
	double beta,zeta,mobi,kappa,phic,gravity;
	parameter (&beta,&zeta,&mobi,&kappa,&phic,&gravity,ex_h,ey_h,ez_h,wt_h,et_h);
	
	FILE *data_2d_t;
	FILE *data_3d_t;
	FILE *data_2d;
	FILE *data_3d;
	FILE *properties;
	FILE *final_2d;
	FILE *final_3d;

	if(myid == 0){
	hipHostMalloc((void**)&c_f_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&m_f_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&b_f_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&p_f_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&u_f_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&v_f_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&w_f_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&a_f_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&xz_f_h ,sizeof(double)*  nx*nz   );
	
	hipHostMalloc((void**)&c_fdo_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&m_fdo_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&b_fdo_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&p_fdo_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&u_fdo_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&v_fdo_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&w_fdo_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&a_fdo_h  ,sizeof(double)*size_final);
	hipHostMalloc((void**)&xz_fdo_h ,sizeof(double)*  nx*nz   );	
	
	hipHostMalloc((void**)&lx ,sizeof(double)*   stepall/2   );
	hipHostMalloc((void**)&lz ,sizeof(double)*   stepall/2   );
	hipHostMalloc((void**)&Re ,sizeof(double)*   stepall/2   );
	initial_macro(c_f_h,m_f_h,b_f_h,p_f_h,u_f_h,v_f_h,w_f_h);
	
	array_2D_do(c_f_h,c_fdo_h);
	array_2D_do(m_f_h,m_fdo_h);
	array_2D_do(b_f_h,b_fdo_h);
	array_2D_do(p_f_h,p_fdo_h);
	array_2D_do(u_f_h,u_fdo_h);
	array_2D_do(v_f_h,v_fdo_h);
	array_2D_do(w_f_h,w_fdo_h);
	array_2D_do(a_f_h,a_fdo_h);
	
	//writing data
	properties = fopen("properties.txt","w");
	if(condition==0){
	double mo=gravity*(rho_l-rho_g)*pow(tau_l,4)*rho_l*rho_l/81.0/pow(sigma,3);
	printf("===============================================================\n");
	fprintf( properties, "Three dimensional droplets - Bubble rising\n");
	fprintf( properties, "Grid size nx=%d, ny=%d, nz=%d\n",nx,ny,nz);
	fprintf( properties, "Radius=%f, Thickness=%f\n",radd, thick);
	fprintf( properties, "Bo=%f\n",bo);
	fprintf( properties, "Mo=%f\n",mo);
	printf ("Bo=%f\n",bo);
	printf ("Mo=%f\n",mo);
	printf("Three dimensional droplets - One Bubble rising\n");
	printf("===============================================================\n");
	}
	else if(condition==1){
	double mo=gravity*(rho_l-rho_g)*pow(tau_l,4)*rho_l*rho_l/81.0/pow(sigma,3);
	printf("===============================================================\n");
	fprintf( properties, "Three dimensional droplets - Bubble rising\n");
	fprintf( properties, "Grid size nx=%d, ny=%d, nz=%d\n",nx,ny,nz);
	fprintf( properties, "Radius=%f, Thickness=%f\n",radd, thick);
	fprintf( properties, "Bo=%f\n",bo);
	fprintf( properties, "Mo=%f\n",mo);
	printf ("Bo=%f\n",bo);
	printf ("Mo=%f\n",mo);
	printf("Three dimensional droplets - Two Bubble rising\n");
	printf("===============================================================\n");
	}
	
	printf("Initializing...");
	fprintf( properties, "Tau_h =%f, Tau_g=%f, Tau_l=%f\n", tau_h,tau_g,tau_l);
	fprintf( properties, "rho_l =%f, rho_g=%f, sigma=%f\n", rho_l,rho_g,sigma);
	fclose(properties);

	data_2d = fopen("data_2d.dat","w");
	fprintf( data_2d, "VARIABLES=\"X\",\"Z\",\"c\",\"u\",\"v\",\"w\",\"p\"\n");
	fprintf( data_2d, "ZONE T=\"STEP=%d\" F=POINT\n",step);
	fprintf( data_2d, "I=%d, J=%d\n", nx,nz);
	j=ny/2;
	for(k=0;k<nz;k++){
	for(i=0;i<nx;i++){
	index=nx*(k*ny+j)+i;
	fprintf( data_2d, "%d\t%d\t%e\t%e\t%e\t%e\t%e\t\n",
	i,k,c_f_h[index],u_f_h[index],v_f_h[index],w_f_h[index],p_f_h[index]);
	}}
	fclose(data_2d);
	
	data_3d = fopen("data_3d.dat","w");
	fprintf( data_3d, "VARIABLES=\"X\",\"Y\",\"Z\",\"c\"\n");
	fprintf( data_3d, "ZONE T=\"STEP=%d\" F=POINT\n",step);
	fprintf( data_3d, "I=%d, J=%d, K=%d\n", nx,ny,nz);
	for(k=0;k<nz;k++){
	for(j=0;j<ny;j++){
	for(i=0;i<nx;i++){
	index=(nx)*(k*(ny)+j)+i;
	fprintf( data_3d, "%d\t%d\t%d\t%e\t\n",
	i,j,k,c_f_h[index]);
	}}}
	fclose(data_3d);
	printf("done\n");
	printf("===============================================================\n");
	printf("Iterating...\n");
	}
	MPI_Barrier(MPI_COMM_WORLD);
	
	//scatter
	iroot = 0;
	MPI_Scatter((void *)&c_fdo_h[0],n_f, MPI_DOUBLE,(void *)&c_d_h[0],n_f, MPI_DOUBLE,iroot,comm);
	MPI_Scatter((void *)&m_fdo_h[0],n_f, MPI_DOUBLE,(void *)&m_d_h[0],n_f, MPI_DOUBLE,iroot,comm);
	MPI_Scatter((void *)&b_fdo_h[0],n_f, MPI_DOUBLE,(void *)&b_d_h[0],n_f, MPI_DOUBLE,iroot,comm);
	MPI_Scatter((void *)&p_fdo_h[0],n_f, MPI_DOUBLE,(void *)&p_d_h[0],n_f, MPI_DOUBLE,iroot,comm);
	MPI_Scatter((void *)&u_fdo_h[0],n_f, MPI_DOUBLE,(void *)&u_d_h[0],n_f, MPI_DOUBLE,iroot,comm);
	MPI_Scatter((void *)&v_fdo_h[0],n_f, MPI_DOUBLE,(void *)&v_d_h[0],n_f, MPI_DOUBLE,iroot,comm);
	MPI_Scatter((void *)&w_fdo_h[0],n_f, MPI_DOUBLE,(void *)&w_d_h[0],n_f, MPI_DOUBLE,iroot,comm);
	MPI_Scatter((void *)&a_fdo_h[0],n_f, MPI_DOUBLE,(void *)&a_d_h[0],n_f, MPI_DOUBLE,iroot,comm);
	MPI_Barrier(MPI_COMM_WORLD);
	
	//memory allocation on gpu
	hipMalloc((void**)&c_d ,sizeof(double)* size_dicom );
	hipMalloc((void**)&m_d ,sizeof(double)* size_dicom );
	hipMalloc((void**)&b_d ,sizeof(double)* size_dicom );
	hipMalloc((void**)&p_d ,sizeof(double)* size_dicom );
	hipMalloc((void**)&u_d ,sizeof(double)* size_dicom );
	hipMalloc((void**)&v_d ,sizeof(double)* size_dicom );
	hipMalloc((void**)&w_d ,sizeof(double)* size_dicom );
	hipMalloc((void**)&a_d ,sizeof(double)* size_dicom );

	hipMalloc((void**)&h   ,sizeof(double)* size_difun );
	hipMalloc((void**)&g   ,sizeof(double)* size_difun );
	hipMalloc((void**)&h_t ,sizeof(double)* size_difun );
	hipMalloc((void**)&g_t ,sizeof(double)* size_difun );
	
	hipMalloc((void**)&t_c ,sizeof(double)* tran_mac_2 );
	hipMalloc((void**)&t_m ,sizeof(double)* tran_mac_2 );
	hipMalloc((void**)&t_b ,sizeof(double)* tran_mac_2 );
	hipMalloc((void**)&t_p ,sizeof(double)* tran_mac_2 );
	hipMalloc((void**)&t_u ,sizeof(double)* tran_mac_1 );
	hipMalloc((void**)&t_v ,sizeof(double)* tran_mac_1 );
	hipMalloc((void**)&t_w ,sizeof(double)* tran_mac_1 );
	hipMalloc((void**)&t_g ,sizeof(double)* tran_difun );
	hipMalloc((void**)&t_h ,sizeof(double)* tran_difun );
	
	hipMalloc((void**)&t_c_x ,sizeof(double)* tran_mac_2_x );
	hipMalloc((void**)&t_m_x ,sizeof(double)* tran_mac_2_x );
	hipMalloc((void**)&t_b_x ,sizeof(double)* tran_mac_2_x );
	hipMalloc((void**)&t_p_x ,sizeof(double)* tran_mac_2_x );
	hipMalloc((void**)&t_u_x ,sizeof(double)* tran_mac_1_x );
	hipMalloc((void**)&t_v_x ,sizeof(double)* tran_mac_1_x );
	hipMalloc((void**)&t_w_x ,sizeof(double)* tran_mac_1_x );
	hipMalloc((void**)&t_g_x ,sizeof(double)* tran_difun_x );
	hipMalloc((void**)&t_h_x ,sizeof(double)* tran_difun_x );
	
	hipMalloc((void**)&gra_c ,sizeof(double)* size_allgr );
	hipMalloc((void**)&gra_m ,sizeof(double)* size_allgr );
	
	hipMalloc((void**)&xz_d,sizeof(double)*(nx/ip+4)*(nz/kp+4));	
	
	MPI_Barrier(MPI_COMM_WORLD);
	//cpu to gpu
	hipMemcpy(c_d, c_d_h, sizeof(double)* size_dicom , hipMemcpyHostToDevice);
	hipMemcpy(m_d, m_d_h, sizeof(double)* size_dicom , hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_d_h, sizeof(double)* size_dicom , hipMemcpyHostToDevice);
	hipMemcpy(p_d, p_d_h, sizeof(double)* size_dicom , hipMemcpyHostToDevice);
	hipMemcpy(u_d, u_d_h, sizeof(double)* size_dicom , hipMemcpyHostToDevice);
	hipMemcpy(v_d, v_d_h, sizeof(double)* size_dicom , hipMemcpyHostToDevice);
	hipMemcpy(w_d, w_d_h, sizeof(double)* size_dicom , hipMemcpyHostToDevice);
	hipMemcpy(a_d, a_d_h, sizeof(double)* size_dicom , hipMemcpyHostToDevice);

	hipMemcpy(t_c, t_c_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice);
	hipMemcpy(t_m, t_m_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice);
	hipMemcpy(t_b, t_b_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice);
	hipMemcpy(t_p, t_p_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice);
	hipMemcpy(t_u, t_u_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice);
	hipMemcpy(t_v, t_v_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice);
	hipMemcpy(t_w, t_w_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice);
	hipMemcpy(t_g, t_g_h, sizeof(double)* tran_difun , hipMemcpyHostToDevice);
	hipMemcpy(t_h, t_h_h, sizeof(double)* tran_difun , hipMemcpyHostToDevice);
	
	hipMemcpy(t_c_x, t_c_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice);
	hipMemcpy(t_m_x, t_m_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice);
	hipMemcpy(t_b_x, t_b_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice);
	hipMemcpy(t_p_x, t_p_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice);
	hipMemcpy(t_u_x, t_u_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice);
	hipMemcpy(t_v_x, t_v_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice);
	hipMemcpy(t_w_x, t_w_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice);
	hipMemcpy(t_g_x, t_g_x_h, sizeof(double)* tran_difun_x , hipMemcpyHostToDevice);
	hipMemcpy(t_h_x, t_h_x_h, sizeof(double)* tran_difun_x , hipMemcpyHostToDevice);
	
	hipMemcpyToSymbol(HIP_SYMBOL(  eex ),  ex_h,   sizeof(double)*q  );
	hipMemcpyToSymbol(HIP_SYMBOL(  eey ),  ey_h,   sizeof(double)*q  );
	hipMemcpyToSymbol(HIP_SYMBOL(  eez ),  ez_h,   sizeof(double)*q  );
	hipMemcpyToSymbol(HIP_SYMBOL(  wwt ),  wt_h,   sizeof(double)*q  );
	hipMemcpyToSymbol(HIP_SYMBOL(  eet ),  et_h,   sizeof(int   )*q  );
	
	MPI_Barrier(MPI_COMM_WORLD);

	int xd=nx/ip; //x decomposition
	int zd=nz/kp; //z decomposition
	
	int grid_t0		=ny*zd;
	int block_t0	=xd;
	int grid_bc		=ny;
	int block_t0_x	=zd-2;
	int grid_in		=ny*(zd-2);
	int grid_in2	=ny*(zd-4);
	
	hipMalloc((void**)&c   ,sizeof(double)* size_dicom );
	hipMalloc((void**)&m   ,sizeof(double)* size_dicom );
	hipMalloc((void**)&b   ,sizeof(double)* size_dicom );
	hipMalloc((void**)&p   ,sizeof(double)* size_dicom );
	hipMalloc((void**)&u   ,sizeof(double)* size_dicom );
	hipMalloc((void**)&v   ,sizeof(double)* size_dicom );
	hipMalloc((void**)&w   ,sizeof(double)* size_dicom );
	hipMalloc((void**)&a   ,sizeof(double)* size_dicom );

	array_do <<<grid_t0 , block_t0>>>( c_d,c );
	array_do <<<grid_t0 , block_t0>>>( m_d,m );
	array_do <<<grid_t0 , block_t0>>>( b_d,b );
	array_do <<<grid_t0 , block_t0>>>( p_d,p );
	array_do <<<grid_t0 , block_t0>>>( u_d,u );
	array_do <<<grid_t0 , block_t0>>>( v_d,v );
	array_do <<<grid_t0 , block_t0>>>( w_d,w );
	array_do <<<grid_t0 , block_t0>>>( a_d,a );
	MPI_Barrier(MPI_COMM_WORLD);

///////////////////////////////////////////////////////////////////////////////////////////
	int num_trans_m_2	=(xd+4)*(ny+4)*2;
	int num_trans_m_1	=(xd+4)*(ny+4)*1;
	int startb			=(xd+4)*( 0 *(ny+4)+0)+0;
	int start			=(xd+4)*( 2 *(ny+4)+0)+0;
	int end				=(xd+4)*( 4 *(ny+4)+0)+0;
	int endb			=(xd+4)*( 6 *(ny+4)+0)+0;
	int startb_1		=(xd+4)*( 0 *(ny+4)+0)+0;
	int start_1			=(xd+4)*( 1 *(ny+4)+0)+0;
	int end_1			=(xd+4)*( 2 *(ny+4)+0)+0;
	int endb_1			=(xd+4)*( 3 *(ny+4)+0)+0;
	int num_trans_d		=(xd+4)*(ny+4)*5;
	int startb_d		=((xd+4)*( 0 *(ny+4)+0)+0)*5;
	int start_d			=((xd+4)*( 1 *(ny+4)+0)+0)*5;
	int end_d			=((xd+4)*( 2 *(ny+4)+0)+0)*5;
	int endb_d			=((xd+4)*( 3 *(ny+4)+0)+0)*5;

	int num_trans_m_2_x	=(ny+4)*(zd+4)*2;
	int num_trans_m_1_x	=(ny+4)*(zd+4)*1;
	int startb_x		=(ny+4)*( 0 *(zd+4)+0)+0;
	int start_x			=(ny+4)*( 2 *(zd+4)+0)+0;
	int end_x			=(ny+4)*( 4 *(zd+4)+0)+0;
	int endb_x			=(ny+4)*( 6 *(zd+4)+0)+0;
	int startb_1_x		=(ny+4)*( 0 *(zd+4)+0)+0;
	int start_1_x		=(ny+4)*( 1 *(zd+4)+0)+0;
	int end_1_x			=(ny+4)*( 2 *(zd+4)+0)+0;
	int endb_1_x		=(ny+4)*( 3 *(zd+4)+0)+0;
	int num_trans_d_x	=(ny+4)*(zd+4)*5;
	int startb_d_x		=((ny+4)*( 0 *(zd+4)+0)+0)*5;
	int start_d_x		=((ny+4)*( 1 *(zd+4)+0)+0)*5;
	int end_d_x			=((ny+4)*( 2 *(zd+4)+0)+0)*5;
	int endb_d_x		=((ny+4)*( 3 *(zd+4)+0)+0)*5;
///////////////////////////////////////////////////////////////////////////////////////////
/* 	checkk <<<grid_t2 , block_t2>>>( c_d,c ); 
	hipMemcpy(c_d_h,c_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	FILE *check;
	if(myid==1){
	check = fopen("check.dat","w");
	fprintf( check, "VARIABLES=\"X\",\"Z\",\"c\"\n");
	fprintf( check, "ZONE T=\"gpu\" F=POINT\n");
	fprintf( check, "I=%d, J=%d\n", nx+4,zd+4);
	j=ny/2;
	for(k=0;k<zd+4;k++){
	for(i=0;i<nx+4;i++){
	index_3d(i,j,k);
	fprintf( check, "%d\t%d\t%e\t\n",
	i,k,c_d_h[index]);
	}}
	fclose(check);
	} */

///////////////////////////////////////////////////////////////////////////////////////////

////y
	boundary_ym		<<< xd+4 , zd+4  >>>( c );
////z
	boundary_zm2	<<< xd+4 , ny+4  >>>( c,t_c );
	hipMemcpy(t_c_h, t_c, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=110;
	MPI_Sendrecv	((void *)&t_c_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_c_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=120;
	MPI_Sendrecv	((void *)&t_c_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_c_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat);

	hipMemcpy(t_c, t_c_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice);
	boundary_zm2_undo	<<< xd+4 , ny+4  >>>( c,t_c );
////x
	boundary_xm2	<<< ny+4 , zd+4  >>>( c,t_c_x );
	hipMemcpy(t_c_x_h, t_c_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=11;
	MPI_Sendrecv	((void *)&t_c_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_c_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=12;
	MPI_Sendrecv	((void *)&t_c_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_c_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat);

	hipMemcpy(t_c_x, t_c_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice);
	boundary_xm2_undo	<<< ny+4 , zd+4  >>>( c,t_c_x );

///////////////////////////////////////////////////////////////////////////////////////////
	chemical   <<<grid_t0, block_t0>>>( c,m,kappa,beta );
//	chemical_b <<<grid_t0, block_t0>>>( c,m,b,kappa,beta,phic );//wettability
///////////////////////////////////////////////////////////////////////////////////////////

////y
	boundary_ym <<< xd+4 , zd+4  >>>( m );
////z
	boundary_zm2<<< xd+4 , ny+4  >>>( m,t_m );
	hipMemcpy(t_m_h, t_m, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=110;
	MPI_Sendrecv	((void *)&t_m_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_m_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=120;
	MPI_Sendrecv	((void *)&t_m_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_m_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpy(t_m, t_m_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice);
	boundary_zm2_undo	<<< xd+4 , ny+4  >>>( m,t_m );
////x
	boundary_xm2<<< ny+4 , zd+4  >>>( m,t_m_x );
	hipMemcpy(t_m_x_h, t_m_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=11;
	MPI_Sendrecv	((void *)&t_m_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_m_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=12;
	MPI_Sendrecv	((void *)&t_m_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_m_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpy(t_m_x, t_m_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice);
	boundary_xm2_undo	<<< ny+4 , zd+4  >>>( m,t_m_x );

	///////////////////////////////////////////////////////////////////////////////////////////

////y
	boundary_ym		<<< xd+4 , zd+4  >>>( b );
////z
	boundary_zm2	<<< xd+4 , ny+4  >>>( b,t_b );
	hipMemcpy(t_b_h, t_b, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=130;
	MPI_Sendrecv	((void *)&t_b_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_b_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=140;
	MPI_Sendrecv	((void *)&t_b_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_b_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpy(t_b, t_b_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice);
	boundary_zm2_undo	<<< xd+4 , ny+4  >>>( b,t_b );
////x
	boundary_xm2	<<< ny+4 , zd+4  >>>( b,t_b_x );
	hipMemcpy(t_b_x_h, t_b_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=13;
	MPI_Sendrecv	((void *)&t_b_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_b_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=14;
	MPI_Sendrecv	((void *)&t_b_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_b_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpy(t_b_x, t_b_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice);
	boundary_xm2_undo	<<< ny+4 , zd+4  >>>( b,t_b_x );

	///////////////////////////////////////////////////////////////////////////////////////////

////y
	boundary_ym		<<< xd+4 , zd+4  >>>( p );
////z
	boundary_zm2	<<< xd+4 , ny+4  >>>( p,t_p );
	hipMemcpy(t_p_h, t_p, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=150;
	MPI_Sendrecv	((void *)&t_p_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_p_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=160;
	MPI_Sendrecv	((void *)&t_p_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_p_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpy(t_p, t_p_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice);
	boundary_zm2_undo	<<< xd+4 , ny+4  >>>( p,t_p );
////x
	boundary_xm2	<<< ny+4 , zd+4  >>>( p,t_p_x );
	hipMemcpy(t_p_x_h, t_p_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=15;
	MPI_Sendrecv	((void *)&t_p_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_p_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=16;
	MPI_Sendrecv	((void *)&t_p_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_p_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpy(t_p_x, t_p_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice);
	boundary_xm2_undo	<<< ny+4 , zd+4  >>>( p,t_p_x );

	///////////////////////////////////////////////////////////////////////////////////////////

////y
	boundary_ym		<<< xd+4 , zd+4  >>>( u );
////z
	boundary_zm1	<<< xd+4 , ny+4  >>>( u,t_u );
	hipMemcpy(t_u_h, t_u, sizeof(double)* tran_mac_1 , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=170;
	MPI_Sendrecv	((void *)&t_u_h[end_1   ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_u_h[startb_1], num_trans_m_1, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=180;
	MPI_Sendrecv	((void *)&t_u_h[start_1 ], num_trans_m_1, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_u_h[endb_1  ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpy(t_u, t_u_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice);
	boundary_zm1_undo	<<< xd+4 , ny+4  >>>( u,t_u );
////x
	boundary_xm1	<<< ny+4 , zd+4  >>>( u,t_u_x );
	hipMemcpy(t_u_x_h, t_u_x, sizeof(double)* tran_mac_1_x , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=17;
	MPI_Sendrecv	((void *)&t_u_x_h[end_1_x   ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_u_x_h[startb_1_x], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=18;
	MPI_Sendrecv	((void *)&t_u_x_h[start_1_x ], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_u_x_h[endb_1_x  ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpy(t_u_x, t_u_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice);
	boundary_xm1_undo	<<< ny+4 , zd+4  >>>( u,t_u_x );

	///////////////////////////////////////////////////////////////////////////////////////////

////y
	boundary_ym		<<< xd+4 , zd+4  >>>( v );
////z
	boundary_zm1	<<< xd+4 , ny+4  >>>( v,t_v );
	hipMemcpy(t_v_h, t_v, sizeof(double)* tran_mac_1 , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=190;
	MPI_Sendrecv	((void *)&t_v_h[end_1   ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_v_h[startb_1], num_trans_m_1, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=200;
	MPI_Sendrecv	((void *)&t_v_h[start_1 ], num_trans_m_1, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_v_h[endb_1  ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpy(t_v, t_v_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice);
	boundary_zm1_undo	<<< xd+4 , ny+4  >>>( v,t_v );
////x
	boundary_xm1	<<< ny+4 , zd+4  >>>( v,t_v_x );
	hipMemcpy(t_v_x_h, t_v_x, sizeof(double)* tran_mac_1_x , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=19;
	MPI_Sendrecv	((void *)&t_v_x_h[end_1_x   ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_v_x_h[startb_1_x], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=20;
	MPI_Sendrecv	((void *)&t_v_x_h[start_1_x ], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_v_x_h[endb_1_x  ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpy(t_v_x, t_v_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice);
	boundary_xm1_undo	<<< ny+4 , zd+4  >>>( v,t_v_x );

	///////////////////////////////////////////////////////////////////////////////////////////

////y
	boundary_ym		<<< xd+4 , zd+4  >>>( w );
////z
	boundary_zm1	<<< xd+4 , ny+4  >>>( w,t_w );
	hipMemcpy(t_w_h, t_w, sizeof(double)* tran_mac_1 , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=210;
	MPI_Sendrecv	((void *)&t_w_h[end_1   ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_w_h[startb_1], num_trans_m_1, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=220;
	MPI_Sendrecv	((void *)&t_w_h[start_1 ], num_trans_m_1, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_w_h[endb_1  ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpy(t_w, t_w_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice);
	boundary_zm1_undo	<<< xd+4 , ny+4  >>>( w,t_w );
////x
	boundary_xm1	<<< ny+4 , zd+4  >>>( w,t_w_x );
	hipMemcpy(t_w_x_h, t_w_x, sizeof(double)* tran_mac_1_x , hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	itag=21;
	MPI_Sendrecv	((void *)&t_w_x_h[end_1_x   ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_w_x_h[startb_1_x], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=22;
	MPI_Sendrecv	((void *)&t_w_x_h[start_1_x ], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_w_x_h[endb_1_x  ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpy(t_w_x, t_w_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice);
	boundary_xm1_undo	<<< ny+4 , zd+4  >>>( w,t_w_x );
	
	MPI_Barrier(MPI_COMM_WORLD);
	
///////////////////////////////////////////////////////////////////////////////////////////
	gradient_cen   <<<grid_t0, block_t0,0>>>(gra_c,c);
	gradient_cen   <<<grid_t0, block_t0,0>>>(gra_m,m);
	hipDeviceSynchronize();
	eq_collision   <<<grid_t0, block_t0  >>>( g,h,c,m,p,gravity,gra_c,gra_m,u,v,w,mobi);
	hipDeviceSynchronize();
	
	hipStream_t  stream0,stream1;
	int leastPriority;
	int greatestPriority;
	hipDeviceGetStreamPriorityRange (&leastPriority,&greatestPriority);
	int priority=greatestPriority;
	hipStreamCreateWithPriority(&stream0,0,priority);
	hipStreamCreate(&stream1);
	//time
	hipEvent_t gpu_start,gpu_start_temp,gpu_stop,gpu_stop_temp;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventCreate(&gpu_start_temp);
	hipEventCreate(&gpu_stop_temp);
	hipEventRecord(gpu_start_temp,0);
	hipEventRecord(gpu_start,0);

///////////////////////////////////////////////////////////////////////////////////////////
//                                        sstart                                         //
///////////////////////////////////////////////////////////////////////////////////////////
	for(step=1;step<=stepall;step++){

	eq_collision_bc    <<< grid_bc    , block_t0   , 0, stream0 >>>( g,h,c,m,p,gravity,gra_c,gra_m,u,v,w,mobi );
	eq_collision_bc_x  <<< grid_bc    , block_t0_x , 0, stream0 >>>( g,h,c,m,p,gravity,gra_c,gra_m,u,v,w,mobi );	
	boundary_yd_bc     <<< xd         , q          , 0, stream0 >>>( g,h );
	boundary_yd_bc_x   <<< zd         , q          , 0, stream0 >>>( g,h );
////z...
	boundary_zd        <<< xd+2       , ny+2       , 0, stream0 >>>( g,t_g );
	boundary_zd        <<< xd+2       , ny+2       , 0, stream0 >>>( h,t_h );
	eq_collision_in    <<< grid_in    , xd-2       , 0, stream1 >>>( g,h,c,m,p,gravity,gra_c,gra_m,u,v,w,mobi );
////...z	
	hipMemcpyAsync(t_g_h, t_g, sizeof(double)*tran_difun , hipMemcpyDeviceToHost,stream0);
	hipMemcpyAsync(t_h_h, t_h, sizeof(double)*tran_difun , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	
	itag=230;
	MPI_Sendrecv	((void *)&t_g_h[end_d   ], num_trans_d, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_g_h[startb_d], num_trans_d, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=231;
	MPI_Sendrecv	((void *)&t_g_h[start_d ], num_trans_d, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_g_h[endb_d  ], num_trans_d, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	itag=232;
	MPI_Sendrecv	((void *)&t_h_h[end_d   ], num_trans_d, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_h_h[startb_d], num_trans_d, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=233;
	MPI_Sendrecv	((void *)&t_h_h[start_d ], num_trans_d, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_h_h[endb_d  ], num_trans_d, MPI_DOUBLE, t_nbr, itag, comm, istat);
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_g, t_g_h, sizeof(double)*tran_difun , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_h, t_h_h, sizeof(double)*tran_difun , hipMemcpyHostToDevice,stream0);
	boundary_zd_undo   <<< xd+2       , ny+2       , 0, stream0 >>>( g,t_g );
	boundary_zd_undo   <<< xd+2       , ny+2       , 0, stream0 >>>( h,t_h );
////x...
	boundary_xd        <<< ny+2       , zd+2       , 0, stream0 >>>( g,t_g_x );
	boundary_xd        <<< ny+2       , zd+2       , 0, stream0 >>>( h,t_h_x );
	boundary_yd_in     <<< xd-2       , zd-2       , 0, stream1 >>>( g,h );
////...x
	hipMemcpyAsync(t_g_x_h, t_g_x, sizeof(double)*tran_difun_x , hipMemcpyDeviceToHost,stream0);
	hipMemcpyAsync(t_h_x_h, t_h_x, sizeof(double)*tran_difun_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	
	itag=23;
	MPI_Sendrecv	((void *)&t_g_x_h[end_d_x   ], num_trans_d_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_g_x_h[startb_d_x], num_trans_d_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=24;
	MPI_Sendrecv	((void *)&t_g_x_h[start_d_x ], num_trans_d_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_g_x_h[endb_d_x  ], num_trans_d_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	itag=25;
	MPI_Sendrecv	((void *)&t_h_x_h[end_d_x   ], num_trans_d_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_h_x_h[startb_d_x], num_trans_d_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=26;
	MPI_Sendrecv	((void *)&t_h_x_h[start_d_x ], num_trans_d_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_h_x_h[endb_d_x  ], num_trans_d_x, MPI_DOUBLE, r_nbr, itag, comm, istat);
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_g_x, t_g_x_h, sizeof(double)*tran_difun_x , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_h_x, t_h_x_h, sizeof(double)*tran_difun_x , hipMemcpyHostToDevice,stream0);
	boundary_xd_undo   <<< ny+2       , zd+2       , 0, stream0 >>>( g,t_g_x );
	boundary_xd_undo   <<< ny+2       , zd+2       , 0, stream0 >>>( h,t_h_x );
///////////////////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();

	macro_h_bc		   <<< grid_bc	  , block_t0   , 0, stream0 >>>( h,h_t,c );
	macro_h_bc_x	   <<< grid_bc    , zd-4	   , 0, stream0 >>>( h,h_t,c );
	boundary_ym_bc	   <<< 1		  , xd		   , 0, stream0 >>>( c );
	boundary_ym_bc_x   <<< 1		  , zd		   , 0, stream0 >>>( c );
////z...
	boundary_zm2       <<< xd+4	      , ny+4       , 0, stream0 >>>( c,t_c );
	macro_h_in		   <<< grid_in2	  , xd-4       , 0, stream1 >>>( h,h_t,c );
////...z	
	hipMemcpyAsync(t_c_h, t_c, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=110;
	MPI_Sendrecv	((void *)&t_c_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_c_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=120;
	MPI_Sendrecv	((void *)&t_c_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_c_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat);
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_c, t_c_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice,stream0);
	boundary_zm2_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( c,t_c );
////x...
	boundary_xm2       <<< ny+4       , zd+4	   , 0, stream0 >>>( c,t_c_x );
	boundary_ym_in     <<< xd-4       ,	zd-4       , 0, stream1 >>>( c );
////...x
	hipMemcpyAsync(t_c_x_h, t_c_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=11;
	MPI_Sendrecv	((void *)&t_c_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_c_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=12;
	MPI_Sendrecv	((void *)&t_c_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_c_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat);
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_c_x, t_c_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice,stream0);
	boundary_xm2_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( c,t_c_x );
///////////////////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();

	chemical_bc		   <<< grid_bc	  , block_t0   , 0, stream0 >>>( c,m,kappa,beta );
	chemical_bc_x	   <<< grid_bc	  , zd-4  	   , 0, stream0 >>>( c,m,kappa,beta );
	boundary_ym_bc	   <<< 1		  , xd		   , 0, stream0 >>>( m );
	boundary_ym_bc_x   <<< 1		  , zd		   , 0, stream0 >>>( m );
////z...
	boundary_zm2	   <<< xd+4	      , ny+4	   , 0, stream0 >>>( m,t_m );
	chemical_in		   <<< grid_in2	  , xd-4  	   , 0, stream1 >>>( c,m,kappa,beta );
////...z	
	hipMemcpyAsync(t_m_h, t_m, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=110;
	MPI_Sendrecv	((void *)&t_m_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_m_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=120;
	MPI_Sendrecv	((void *)&t_m_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_m_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat);
					
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_m, t_m_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice,stream0);
	boundary_zm2_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( m,t_m );
////x...	
	boundary_xm2	   <<< ny+4       , zd+4	   , 0, stream0 >>>( m,t_m_x );
	boundary_ym_in	   <<< xd-4		  , zd-4	   , 0, stream1 >>>( m );	
	gradient_cen	   <<< grid_t0	  , block_t0   , 0, stream1 >>>( gra_c,c );
////...x
	hipMemcpyAsync(t_m_x_h, t_m_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=11;
	MPI_Sendrecv	((void *)&t_m_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_m_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=12;
	MPI_Sendrecv	((void *)&t_m_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_m_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat);
					
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_m_x, t_m_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice,stream0);
	boundary_xm2_undo  <<< ny+4       , zd+4	   , 0, stream0 >>>( m,t_m_x );
///////////////////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();

	gradient_cen	   <<< grid_t0	  , block_t0                >>>( gra_m,m );
	macro_g_bc		   <<< grid_bc 	  , block_t0   , 0, stream0 >>>( g,g_t,c,m,p,gra_c,gra_m,u,v,w );
	macro_g_bc_x	   <<< grid_bc	  , zd-4       , 0, stream0 >>>( g,g_t,c,m,p,gra_c,gra_m,u,v,w );
////y bc	
	boundary_ym_bc	   <<< 1	      , xd         , 0, stream0 >>>( u );
	boundary_ym_bc	   <<< 1	      , xd         , 0, stream0 >>>( v );
	boundary_ym_bc	   <<< 1	      , xd         , 0, stream0 >>>( w );
	boundary_ym_bc	   <<< 1	      , xd         , 0, stream0 >>>( p );

	boundary_ym_bc_x   <<< 1	      , zd         , 0, stream0 >>>( u );
	boundary_ym_bc_x   <<< 1	      , zd         , 0, stream0 >>>( v );
	boundary_ym_bc_x   <<< 1	      , zd         , 0, stream0 >>>( w );
	boundary_ym_bc_x   <<< 1	      , zd         , 0, stream0 >>>( p );
////z...
	boundary_zm2	   <<< xd+4       , ny+4	   , 0, stream0 >>>( p,t_p );
	boundary_zm1	   <<< xd+4       , ny+4	   , 0, stream0 >>>( u,t_u );
	boundary_zm1	   <<< xd+4       , ny+4	   , 0, stream0 >>>( v,t_v );
	boundary_zm1	   <<< xd+4       , ny+4	   , 0, stream0 >>>( w,t_w );
	macro_g_in		   <<< grid_in2	  , xd-4       , 0, stream1 >>>( g,g_t,c,m,p,gra_c,gra_m,u,v,w);
////...z
	hipMemcpyAsync(t_p_h, t_p, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=150;
	MPI_Sendrecv	((void *)&t_p_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_p_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=160;
	MPI_Sendrecv	((void *)&t_p_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_p_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat); 

	hipMemcpyAsync(t_u_h, t_u, sizeof(double)* tran_mac_1 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=170;
	MPI_Sendrecv	((void *)&t_u_h[end_1   ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_u_h[startb_1], num_trans_m_1, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=180;
	MPI_Sendrecv	((void *)&t_u_h[start_1 ], num_trans_m_1, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_u_h[endb_1  ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpyAsync(t_v_h, t_v, sizeof(double)* tran_mac_1 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=190;
	MPI_Sendrecv	((void *)&t_v_h[end_1   ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_v_h[startb_1], num_trans_m_1, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=200;
	MPI_Sendrecv	((void *)&t_v_h[start_1 ], num_trans_m_1, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_v_h[endb_1  ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpyAsync(t_w_h, t_w, sizeof(double)* tran_mac_1 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=210;
	MPI_Sendrecv	((void *)&t_w_h[end_1   ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_w_h[startb_1], num_trans_m_1, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=220;
	MPI_Sendrecv	((void *)&t_w_h[start_1 ], num_trans_m_1, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_w_h[endb_1  ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_p, t_p_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_u, t_u_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_v, t_v_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_w, t_w_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice,stream0);
	
	hipStreamSynchronize(stream0);
	boundary_zm2_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( p,t_p );
	boundary_zm1_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( u,t_u );
	boundary_zm1_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( v,t_v );
	boundary_zm1_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( w,t_w );
////x...
	boundary_xm2	   <<< ny+4       , zd+4       , 0, stream0 >>>( p,t_p_x );
	boundary_xm1	   <<< ny+4       , zd+4       , 0, stream0 >>>( u,t_u_x );
	boundary_xm1	   <<< ny+4       , zd+4       , 0, stream0 >>>( v,t_v_x );
	boundary_xm1	   <<< ny+4       , zd+4       , 0, stream0 >>>( w,t_w_x );
	boundary_ym_in	   <<< xd-4       ,	zd-4       , 0, stream1 >>>( p );
	boundary_ym_in	   <<< xd-4       ,	zd-4       , 0, stream1 >>>( u );
	boundary_ym_in	   <<< xd-4       ,	zd-4       , 0, stream1 >>>( v );
	boundary_ym_in	   <<< xd-4       ,	zd-4       , 0, stream1 >>>( w );
////...x
	hipMemcpyAsync(t_p_x_h, t_p_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=15;
	MPI_Sendrecv	((void *)&t_p_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_p_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=16;
	MPI_Sendrecv	((void *)&t_p_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_p_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 

	hipMemcpyAsync(t_u_x_h, t_u_x, sizeof(double)* tran_mac_1_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=17;
	MPI_Sendrecv	((void *)&t_u_x_h[end_1_x   ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_u_x_h[startb_1_x], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=18;
	MPI_Sendrecv	((void *)&t_u_x_h[start_1_x ], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_u_x_h[endb_1_x  ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpyAsync(t_v_x_h, t_v_x, sizeof(double)* tran_mac_1_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=19;
	MPI_Sendrecv	((void *)&t_v_x_h[end_1_x   ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_v_x_h[startb_1_x], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=20;
	MPI_Sendrecv	((void *)&t_v_x_h[start_1_x ], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_v_x_h[endb_1_x  ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpyAsync(t_w_x_h, t_w_x, sizeof(double)* tran_mac_1_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=21;
	MPI_Sendrecv	((void *)&t_w_x_h[end_1_x   ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_w_x_h[startb_1_x], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=22;
	MPI_Sendrecv	((void *)&t_w_x_h[start_1_x ], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_w_x_h[endb_1_x  ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_p_x, t_p_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_u_x, t_u_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_v_x, t_v_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_w_x, t_w_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice,stream0);
	
	hipStreamSynchronize(stream0);
	boundary_xm2_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( p,t_p_x );
	boundary_xm1_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( u,t_u_x );
	boundary_xm1_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( v,t_v_x );
	boundary_xm1_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( w,t_w_x );
///////////////////////////////////////////////////////////////////////////////////////////
//                                    nnext time step                                    //
///////////////////////////////////////////////////////////////////////////////////////////
	step=step+1;
	hipDeviceSynchronize();
		
	eq_collision_bc    <<< grid_bc    , block_t0   , 0, stream0 >>>( g_t,h_t,c,m,p,gravity,gra_c,gra_m,u,v,w,mobi );
	eq_collision_bc_x  <<< grid_bc    , block_t0_x , 0, stream0 >>>( g_t,h_t,c,m,p,gravity,gra_c,gra_m,u,v,w,mobi );	
	boundary_yd_bc     <<< xd         , q          , 0, stream0 >>>( g_t,h_t );
	boundary_yd_bc_x   <<< zd         , q          , 0, stream0 >>>( g_t,h_t );
////z...
	boundary_zd        <<< xd+2       , ny+2       , 0, stream0 >>>( g_t,t_g );
	boundary_zd        <<< xd+2       , ny+2       , 0, stream0 >>>( h_t,t_h );
	eq_collision_in    <<< grid_in    , xd-2       , 0, stream1 >>>( g_t,h_t,c,m,p,gravity,gra_c,gra_m,u,v,w,mobi );
////...z	
	hipMemcpyAsync(t_g_h, t_g, sizeof(double)*tran_difun , hipMemcpyDeviceToHost,stream0);
	hipMemcpyAsync(t_h_h, t_h, sizeof(double)*tran_difun , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	
	itag=230;
	MPI_Sendrecv	((void *)&t_g_h[end_d   ], num_trans_d, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_g_h[startb_d], num_trans_d, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=231;
	MPI_Sendrecv	((void *)&t_g_h[start_d ], num_trans_d, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_g_h[endb_d  ], num_trans_d, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	itag=232;
	MPI_Sendrecv	((void *)&t_h_h[end_d   ], num_trans_d, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_h_h[startb_d], num_trans_d, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=233;
	MPI_Sendrecv	((void *)&t_h_h[start_d ], num_trans_d, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_h_h[endb_d  ], num_trans_d, MPI_DOUBLE, t_nbr, itag, comm, istat);
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_g, t_g_h, sizeof(double)*tran_difun , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_h, t_h_h, sizeof(double)*tran_difun , hipMemcpyHostToDevice,stream0);
	boundary_zd_undo   <<< xd+2       , ny+2       , 0, stream0 >>>( g_t,t_g );
	boundary_zd_undo   <<< xd+2       , ny+2       , 0, stream0 >>>( h_t,t_h );
////x...
	boundary_xd        <<< ny+2       , zd+2       , 0, stream0 >>>( g_t,t_g_x );
	boundary_xd        <<< ny+2       , zd+2       , 0, stream0 >>>( h_t,t_h_x );
	boundary_yd_in     <<< xd-2       , zd-2       , 0, stream1 >>>( g_t,h_t );
////...x
	hipMemcpyAsync(t_g_x_h, t_g_x, sizeof(double)*tran_difun_x , hipMemcpyDeviceToHost,stream0);
	hipMemcpyAsync(t_h_x_h, t_h_x, sizeof(double)*tran_difun_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	
	itag=23;
	MPI_Sendrecv	((void *)&t_g_x_h[end_d_x   ], num_trans_d_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_g_x_h[startb_d_x], num_trans_d_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=24;
	MPI_Sendrecv	((void *)&t_g_x_h[start_d_x ], num_trans_d_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_g_x_h[endb_d_x  ], num_trans_d_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	itag=25;
	MPI_Sendrecv	((void *)&t_h_x_h[end_d_x   ], num_trans_d_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_h_x_h[startb_d_x], num_trans_d_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=26;
	MPI_Sendrecv	((void *)&t_h_x_h[start_d_x ], num_trans_d_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_h_x_h[endb_d_x  ], num_trans_d_x, MPI_DOUBLE, r_nbr, itag, comm, istat);
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_g_x, t_g_x_h, sizeof(double)*tran_difun_x , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_h_x, t_h_x_h, sizeof(double)*tran_difun_x , hipMemcpyHostToDevice,stream0);
	boundary_xd_undo   <<< ny+2       , zd+2       , 0, stream0 >>>( g_t,t_g_x );
	boundary_xd_undo   <<< ny+2       , zd+2       , 0, stream0 >>>( h_t,t_h_x );
///////////////////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();

	macro_h_bc		   <<< grid_bc	  , block_t0   , 0, stream0 >>>( h_t,h,c );
	macro_h_bc_x	   <<< grid_bc    , zd-4	   , 0, stream0 >>>( h_t,h,c );
	boundary_ym_bc	   <<< 1		  , xd		   , 0, stream0 >>>( c );
	boundary_ym_bc_x   <<< 1		  , zd		   , 0, stream0 >>>( c );
////z...
	boundary_zm2       <<< xd+4	      , ny+4       , 0, stream0 >>>( c,t_c );
	macro_h_in		   <<< grid_in2	  , xd-4       , 0, stream1 >>>( h_t,h,c );
////...z	
	hipMemcpyAsync(t_c_h, t_c, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=110;
	MPI_Sendrecv	((void *)&t_c_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_c_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=120;
	MPI_Sendrecv	((void *)&t_c_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_c_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat);
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_c, t_c_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice,stream0);
	boundary_zm2_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( c,t_c );
////x...
	boundary_xm2       <<< ny+4       , zd+4	   , 0, stream0 >>>( c,t_c_x );
	boundary_ym_in     <<< xd-4       ,	zd-4       , 0, stream1 >>>( c );
////...x
	hipMemcpyAsync(t_c_x_h, t_c_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=11;
	MPI_Sendrecv	((void *)&t_c_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_c_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=12;
	MPI_Sendrecv	((void *)&t_c_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_c_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat);
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_c_x, t_c_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice,stream0);
	boundary_xm2_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( c,t_c_x );
///////////////////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();

	chemical_bc		   <<< grid_bc	  , block_t0   , 0, stream0 >>>( c,m,kappa,beta );
	chemical_bc_x	   <<< grid_bc	  , zd-4  	   , 0, stream0 >>>( c,m,kappa,beta );
	boundary_ym_bc	   <<< 1		  , xd		   , 0, stream0 >>>( m );
	boundary_ym_bc_x   <<< 1		  , zd		   , 0, stream0 >>>( m );
////z...
	boundary_zm2	   <<< xd+4	      , ny+4	   , 0, stream0 >>>( m,t_m );
	chemical_in		   <<< grid_in2	  , xd-4  	   , 0, stream1 >>>( c,m,kappa,beta );
////...z	
	hipMemcpyAsync(t_m_h, t_m, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=110;
	MPI_Sendrecv	((void *)&t_m_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_m_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=120;
	MPI_Sendrecv	((void *)&t_m_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_m_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat);
					
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_m, t_m_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice,stream0);
	boundary_zm2_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( m,t_m );
////x...	
	boundary_xm2	   <<< ny+4       , zd+4	   , 0, stream0 >>>( m,t_m_x );
	boundary_ym_in	   <<< xd-4		  , zd-4	   , 0, stream1 >>>( m );	
	gradient_cen	   <<< grid_t0	  , block_t0   , 0, stream1 >>>( gra_c,c );
////...x
	hipMemcpyAsync(t_m_x_h, t_m_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=11;
	MPI_Sendrecv	((void *)&t_m_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_m_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=12;
	MPI_Sendrecv	((void *)&t_m_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_m_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat);
					
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_m_x, t_m_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice,stream0);
	boundary_xm2_undo  <<< ny+4       , zd+4	   , 0, stream0 >>>( m,t_m_x );
///////////////////////////////////////////////////////////////////////////////////////////
	hipDeviceSynchronize();

	gradient_cen	   <<< grid_t0	  , block_t0                >>>( gra_m,m );
	macro_g_bc		   <<< grid_bc 	  , block_t0   , 0, stream0 >>>( g_t,g,c,m,p,gra_c,gra_m,u,v,w );
	macro_g_bc_x	   <<< grid_bc	  , zd-4       , 0, stream0 >>>( g_t,g,c,m,p,gra_c,gra_m,u,v,w );
////y bc	
	boundary_ym_bc	   <<< 1	      , xd         , 0, stream0 >>>( u );
	boundary_ym_bc	   <<< 1	      , xd         , 0, stream0 >>>( v );
	boundary_ym_bc	   <<< 1	      , xd         , 0, stream0 >>>( w );
	boundary_ym_bc	   <<< 1	      , xd         , 0, stream0 >>>( p );

	boundary_ym_bc_x   <<< 1	      , zd         , 0, stream0 >>>( u );
	boundary_ym_bc_x   <<< 1	      , zd         , 0, stream0 >>>( v );
	boundary_ym_bc_x   <<< 1	      , zd         , 0, stream0 >>>( w );
	boundary_ym_bc_x   <<< 1	      , zd         , 0, stream0 >>>( p );
////z...
	boundary_zm2	   <<< xd+4       , ny+4	   , 0, stream0 >>>( p,t_p );
	boundary_zm1	   <<< xd+4       , ny+4	   , 0, stream0 >>>( u,t_u );
	boundary_zm1	   <<< xd+4       , ny+4	   , 0, stream0 >>>( v,t_v );
	boundary_zm1	   <<< xd+4       , ny+4	   , 0, stream0 >>>( w,t_w );
	macro_g_in		   <<< grid_in2	  , xd-4       , 0, stream1 >>>( g_t,g,c,m,p,gra_c,gra_m,u,v,w);
////...z
	hipMemcpyAsync(t_p_h, t_p, sizeof(double)* tran_mac_2 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=150;
	MPI_Sendrecv	((void *)&t_p_h[end   ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_p_h[startb], num_trans_m_2, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=160;
	MPI_Sendrecv	((void *)&t_p_h[start ], num_trans_m_2, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_p_h[endb  ], num_trans_m_2, MPI_DOUBLE, t_nbr, itag, comm, istat); 

	hipMemcpyAsync(t_u_h, t_u, sizeof(double)* tran_mac_1 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=170;
	MPI_Sendrecv	((void *)&t_u_h[end_1   ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_u_h[startb_1], num_trans_m_1, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=180;
	MPI_Sendrecv	((void *)&t_u_h[start_1 ], num_trans_m_1, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_u_h[endb_1  ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpyAsync(t_v_h, t_v, sizeof(double)* tran_mac_1 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=190;
	MPI_Sendrecv	((void *)&t_v_h[end_1   ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_v_h[startb_1], num_trans_m_1, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=200;
	MPI_Sendrecv	((void *)&t_v_h[start_1 ], num_trans_m_1, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_v_h[endb_1  ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipMemcpyAsync(t_w_h, t_w, sizeof(double)* tran_mac_1 , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=210;
	MPI_Sendrecv	((void *)&t_w_h[end_1   ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag,
					( void *)&t_w_h[startb_1], num_trans_m_1, MPI_DOUBLE, b_nbr, itag, comm, istat);
	itag=220;
	MPI_Sendrecv	((void *)&t_w_h[start_1 ], num_trans_m_1, MPI_DOUBLE, b_nbr, itag,
					( void *)&t_w_h[endb_1  ], num_trans_m_1, MPI_DOUBLE, t_nbr, itag, comm, istat); 
	
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_p, t_p_h, sizeof(double)* tran_mac_2 , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_u, t_u_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_v, t_v_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_w, t_w_h, sizeof(double)* tran_mac_1 , hipMemcpyHostToDevice,stream0);
	
	hipStreamSynchronize(stream0);
	boundary_zm2_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( p,t_p );
	boundary_zm1_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( u,t_u );
	boundary_zm1_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( v,t_v );
	boundary_zm1_undo  <<< xd+4       , ny+4       , 0, stream0 >>>( w,t_w );
////x...
	boundary_xm2	   <<< ny+4       , zd+4       , 0, stream0 >>>( p,t_p_x );
	boundary_xm1	   <<< ny+4       , zd+4       , 0, stream0 >>>( u,t_u_x );
	boundary_xm1	   <<< ny+4       , zd+4       , 0, stream0 >>>( v,t_v_x );
	boundary_xm1	   <<< ny+4       , zd+4       , 0, stream0 >>>( w,t_w_x );
	boundary_ym_in	   <<< xd-4       ,	zd-4       , 0, stream1 >>>( p );
	boundary_ym_in	   <<< xd-4       ,	zd-4       , 0, stream1 >>>( u );
	boundary_ym_in	   <<< xd-4       ,	zd-4       , 0, stream1 >>>( v );
	boundary_ym_in	   <<< xd-4       ,	zd-4       , 0, stream1 >>>( w );
////...x
	hipMemcpyAsync(t_p_x_h, t_p_x, sizeof(double)* tran_mac_2_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=15;
	MPI_Sendrecv	((void *)&t_p_x_h[end_x   ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_p_x_h[startb_x], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=16;
	MPI_Sendrecv	((void *)&t_p_x_h[start_x ], num_trans_m_2_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_p_x_h[endb_x  ], num_trans_m_2_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 

	hipMemcpyAsync(t_u_x_h, t_u_x, sizeof(double)* tran_mac_1_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=17;
	MPI_Sendrecv	((void *)&t_u_x_h[end_1_x   ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_u_x_h[startb_1_x], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=18;
	MPI_Sendrecv	((void *)&t_u_x_h[start_1_x ], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_u_x_h[endb_1_x  ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpyAsync(t_v_x_h, t_v_x, sizeof(double)* tran_mac_1_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=19;
	MPI_Sendrecv	((void *)&t_v_x_h[end_1_x   ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_v_x_h[startb_1_x], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=20;
	MPI_Sendrecv	((void *)&t_v_x_h[start_1_x ], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_v_x_h[endb_1_x  ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipMemcpyAsync(t_w_x_h, t_w_x, sizeof(double)* tran_mac_1_x , hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
	itag=21;
	MPI_Sendrecv	((void *)&t_w_x_h[end_1_x   ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag,
					( void *)&t_w_x_h[startb_1_x], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag, comm, istat);
	itag=22;
	MPI_Sendrecv	((void *)&t_w_x_h[start_1_x ], num_trans_m_1_x, MPI_DOUBLE, l_nbr, itag,
					( void *)&t_w_x_h[endb_1_x  ], num_trans_m_1_x, MPI_DOUBLE, r_nbr, itag, comm, istat); 
	
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(t_p_x, t_p_x_h, sizeof(double)* tran_mac_2_x , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_u_x, t_u_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_v_x, t_v_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(t_w_x, t_w_x_h, sizeof(double)* tran_mac_1_x , hipMemcpyHostToDevice,stream0);
	
	hipStreamSynchronize(stream0);
	boundary_xm2_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( p,t_p_x );
	boundary_xm1_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( u,t_u_x );
	boundary_xm1_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( v,t_v_x );
	boundary_xm1_undo  <<< ny+4       , zd+4       , 0, stream0 >>>( w,t_w_x );
	
	if(condition == 0){
	array_undo <<<grid_t0 , block_t0>>>( c_d,c );
	array_undo <<<grid_t0 , block_t0>>>( w_d,w );
	MPI_Barrier(MPI_COMM_WORLD);
	hipMemcpy(c_d_h,c_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(w_d_h,w_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	MPI_Gather((void *)&c_d_h[0], n_f, MPI_DOUBLE,(void *)&c_f_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&w_d_h[0], n_f, MPI_DOUBLE,(void *)&w_f_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Barrier(MPI_COMM_WORLD);
	if(myid==0){
	double	maxw;
	max_w(c_f_h,w_f_h,&maxw);
	Reynolds_Time( maxw, Re, step );
	}}
	
	if(step%iprint ==0){
	
	p_real	   <<<grid_t0 , block_t0>>>(c,p,a,beta,kappa,gra_c);
	
	array_undo <<<grid_t0 , block_t0>>>( c_d,c );
	array_undo <<<grid_t0 , block_t0>>>( m_d,m );
	array_undo <<<grid_t0 , block_t0>>>( b_d,b );
	array_undo <<<grid_t0 , block_t0>>>( p_d,p );
	array_undo <<<grid_t0 , block_t0>>>( u_d,u );
	array_undo <<<grid_t0 , block_t0>>>( v_d,v );
	array_undo <<<grid_t0 , block_t0>>>( w_d,w );
	array_undo <<<grid_t0 , block_t0>>>( a_d,a );
	MPI_Barrier(MPI_COMM_WORLD);
	
	hipMemcpy(c_d_h,c_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(m_d_h,m_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(b_d_h,b_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(p_d_h,p_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(u_d_h,u_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(v_d_h,v_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(w_d_h,w_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(a_d_h,a_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);
	
	MPI_Gather((void *)&c_d_h[0], n_f, MPI_DOUBLE,(void *)&c_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&m_d_h[0], n_f, MPI_DOUBLE,(void *)&m_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&b_d_h[0], n_f, MPI_DOUBLE,(void *)&b_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&p_d_h[0], n_f, MPI_DOUBLE,(void *)&p_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&u_d_h[0], n_f, MPI_DOUBLE,(void *)&u_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&v_d_h[0], n_f, MPI_DOUBLE,(void *)&v_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&w_d_h[0], n_f, MPI_DOUBLE,(void *)&w_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&a_d_h[0], n_f, MPI_DOUBLE,(void *)&a_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Barrier(MPI_COMM_WORLD);
	
	if(myid==0){
		
	array_2D_undo(c_f_h,c_fdo_h);
	array_2D_undo(m_f_h,m_fdo_h);
	array_2D_undo(b_f_h,b_fdo_h);
	array_2D_undo(p_f_h,p_fdo_h);
	array_2D_undo(u_f_h,u_fdo_h);
	array_2D_undo(v_f_h,v_fdo_h);
	array_2D_undo(w_f_h,w_fdo_h);
	array_2D_undo(a_f_h,a_fdo_h);
	
	printf("step=%d\n",step);
	hipEventRecord(gpu_stop_temp,0);
	hipEventSynchronize(gpu_stop_temp);
	float cudatime_temp;
	hipEventElapsedTime(&cudatime_temp,gpu_start_temp,gpu_stop_temp);
	cudatime_temp=cudatime_temp/1000.0;//unit sec
	int remain_time=(int)(cudatime_temp/iprint*(stepall-step));
	printf("time remaining: %d hr,%d min,%d sec\n",(int)remain_time/3600,(int)(remain_time%3600)/60,(int)remain_time%60);
	int indexx;
	printf("c max=%lf\n",maxvalue(c_f_h,&indexx));
	printf("c min=%lf\n",minvalue(c_f_h,&indexx));
	printf("p max=%e\n" ,maxvalue(p_f_h,&indexx));
	printf("u max=%e\n" ,maxvalue(u_f_h,&indexx));
	printf("v max=%e\n" ,maxvalue(v_f_h,&indexx));
	printf("w max=%e\n" ,maxvalue(w_f_h,&indexx));

	data_2d = fopen("data_2d.dat","a");
	fprintf( data_2d, "VARIABLES=\"X\",\"Z\",\"c\",\"u\",\"v\",\"w\",\"p\"\n");
	fprintf( data_2d, "ZONE T=\"STEP=%d\" F=POINT\n",step);
	fprintf( data_2d, "I=%d, J=%d\n", nx,nz);
	j=ny/2;
	for(k=0;k<nz;k++){
	for(i=0;i<nx;i++){
	index=nx*(k*ny+j)+i;
	fprintf( data_2d, "%d\t%d\t%e\t%e\t%e\t%e\t%e\t\n",
	i,k,c_f_h[index],u_f_h[index],v_f_h[index],w_f_h[index],p_f_h[index]);
	}}
	fclose(data_2d);
	
	data_2d_t = fopen("data_2d_t.dat","w");
	fprintf( data_2d_t, "VARIABLES=\"X\",\"Z\",\"c\",\"u\",\"v\",\"w\",\"p\"\n");
	fprintf( data_2d_t, "ZONE T=\"STEP=%d\" F=POINT\n",step);
	fprintf( data_2d_t, "I=%d, J=%d\n", nx,nz);
	j=ny/2;
	for(k=0;k<nz;k++){
	for(i=0;i<nx;i++){
	index=nx*(k*ny+j)+i;
	fprintf( data_2d_t, "%d\t%d\t%e\t%e\t%e\t%e\t%e\t\n",
	i,k,c_f_h[index],u_f_h[index],v_f_h[index],w_f_h[index],p_f_h[index]);
	}}
	fclose(data_2d_t);
	
	
	if(step%idata_3d ==0){
	data_3d = fopen("data_3d.dat","a");
	fprintf( data_3d, "VARIABLES=\"X\",\"Y\",\"Z\",\"c\"\n");
	fprintf( data_3d, "ZONE T=\"STEP=%d\" F=POINT\n",step);
	fprintf( data_3d, "I=%d, J=%d, K=%d\n", nx,ny,nz);
	for(k=0;k<nz;k++){
	for(j=0;j<ny;j++){
	for(i=0;i<nx;i++){
	index=(nx)*(k*(ny)+j)+i;
	fprintf( data_3d, "%d\t%d\t%d\t%e\t\n",
	i,j,k,c_f_h[index]);
	}}}
	fclose(data_3d);
	
	data_3d_t = fopen("data_3d_t.dat","w");
	fprintf( data_3d_t, "VARIABLES=\"X\",\"Y\",\"Z\",\"c\"\n");
	fprintf( data_3d_t, "ZONE T=\"STEP=%d\" F=POINT\n",step);
	fprintf( data_3d_t, "I=%d, J=%d, K=%d\n", nx,ny,nz);
	for(k=0;k<nz;k++){
	for(j=0;j<ny;j++){
	for(i=0;i<nx;i++){
	index=(nx)*(k*(ny)+j)+i;
	fprintf( data_3d_t, "%d\t%d\t%d\t%e\t\n",
	i,j,k,c_f_h[index]);
	}}}
	fclose(data_3d_t);	
	}
	printf("===============================================================\n");
	}
	hipEventRecord(gpu_start_temp,0);
	}
	} 
///////////////////////////////////////////////////////////////////////////////////////////
//                                        eend                                           //
///////////////////////////////////////////////////////////////////////////////////////////
	MPI_Barrier(MPI_COMM_WORLD);
	hipEventRecord(gpu_stop,0);
	hipEventSynchronize(gpu_stop);
	float cudatime;
	if(myid==0){
	printf("===============================================================\n");
	printf("Iteration terminated!\n");
	hipEventElapsedTime(&cudatime,gpu_start,gpu_stop);
	printf("GPU total time = %f ms\n",cudatime); //unit = ms
	printf("mlups=%lf \n",(double)(nx*ny*nz)*stepall*pow(10.0,-6.0)/(cudatime/1000.0));
	printf("===============================================================\n");
	}
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);
	
	array_undo <<<grid_t0 , block_t0>>>( c_d,c );
	array_undo <<<grid_t0 , block_t0>>>( m_d,m );
	array_undo <<<grid_t0 , block_t0>>>( b_d,b );
	array_undo <<<grid_t0 , block_t0>>>( p_d,p );
	array_undo <<<grid_t0 , block_t0>>>( u_d,u );
	array_undo <<<grid_t0 , block_t0>>>( v_d,v );
	array_undo <<<grid_t0 , block_t0>>>( w_d,w );
	array_undo <<<grid_t0 , block_t0>>>( a_d,a );
	MPI_Barrier(MPI_COMM_WORLD);
	
	hipMemcpy(c_d_h,c_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(m_d_h,m_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(b_d_h,b_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(p_d_h,p_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(u_d_h,u_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(v_d_h,v_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(w_d_h,w_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	hipMemcpy(a_d_h,a_d,sizeof(double)*size_dicom,hipMemcpyDeviceToHost);
	MPI_Barrier(MPI_COMM_WORLD);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	MPI_Gather((void *)&c_d_h[0], n_f, MPI_DOUBLE,(void *)&c_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&m_d_h[0], n_f, MPI_DOUBLE,(void *)&m_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&b_d_h[0], n_f, MPI_DOUBLE,(void *)&b_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&p_d_h[0], n_f, MPI_DOUBLE,(void *)&p_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&u_d_h[0], n_f, MPI_DOUBLE,(void *)&u_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&v_d_h[0], n_f, MPI_DOUBLE,(void *)&v_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&w_d_h[0], n_f, MPI_DOUBLE,(void *)&w_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Gather((void *)&a_d_h[0], n_f, MPI_DOUBLE,(void *)&a_fdo_h[0],   n_f, MPI_DOUBLE,iroot,comm);
	MPI_Barrier(MPI_COMM_WORLD);
	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	if(myid==0){
		
	array_2D_undo(c_f_h,c_fdo_h);
	array_2D_undo(m_f_h,m_fdo_h);
	array_2D_undo(b_f_h,b_fdo_h);
	array_2D_undo(p_f_h,p_fdo_h);
	array_2D_undo(u_f_h,u_fdo_h);
	array_2D_undo(v_f_h,v_fdo_h);
	array_2D_undo(w_f_h,w_fdo_h);
	array_2D_undo(a_f_h,a_fdo_h);
	
	final_2d = fopen("final_2d.dat","w");
	fprintf( final_2d, "VARIABLES=\"X\",\"Z\",\"c\",\"u\",\"v\",\"w\",\"p\",\"p_real\"\n");
	fprintf( final_2d, "ZONE T=\"STEP=%d\" F=POINT\n",step);
	fprintf( final_2d, "I=%d, J=%d\n", nx,nz);
	j=ny/2;
	for(k=0;k<nz;k++){
	for(i=0;i<nx;i++){
	index=nx*(k*ny+j)+i;
	fprintf( final_2d, "%d\t%d\t%e\t%e\t%e\t%e\t%e\t%e\t\n",
	i,k,c_f_h[index],u_f_h[index],v_f_h[index],w_f_h[index],p_f_h[index],a_f_h[index]);
	}}
	fclose(final_2d);
	
	final_3d = fopen("final_3d.dat","w");
	fprintf( final_3d, "VARIABLES=\"X\",\"Y\",\"Z\",\"c\",\"u\",\"v\",\"w\",\"p\",\"p_real\"\n");
	fprintf( final_3d, "ZONE T=\"STEP=%d\" F=POINT\n",step);
	fprintf( final_3d, "I=%d, J=%d, K=%d\n", nx,ny,nz);
	for(k=0;k<nz;k++){
	for(j=0;j<ny;j++){
	for(i=0;i<nx;i++){
	index=(nx)*(k*(ny)+j)+i;
	fprintf( final_3d, "%d\t%d\t%d\t%e\t%e\t%e\t%e\t%e\t%e\t\n",
	i,j,k,c_f_h[index],u_f_h[index],v_f_h[index],w_f_h[index],p_f_h[index],a_f_h[index]);
	}}}
	fclose(final_3d);

	properties = fopen("properties.txt","a");
	fprintf( properties,"MLUPS =%e\n",(double)(nx*ny*nz)*stepall*pow(10.0,-6.0)/(cudatime/1000.0));
 	if(condition == 0){
	FILE *Reynolds;
	double T=sqrt(radd*2/gravity);
	Reynolds = fopen("Reynolds.dat","w");
	fprintf( Reynolds, "VARIABLES=\"T\",\"Reynolds\"\n");
	fprintf( Reynolds, "ZONE T=\"Reynolds\" F=POINT\n");
	fprintf( Reynolds, "I=%d\n", stepall/2);
 	for(i=0;i<stepall/2;i++){
	fprintf( Reynolds, "%e\t%e\n",(double)2*(i+1)/T,Re[i]);}
	fclose ( Reynolds);
	}
	}
	// Free memory
	hipHostFree(  c_d_h  );
	hipHostFree(  m_d_h  );
	hipHostFree(  b_d_h  );
	hipHostFree(  p_d_h  );
	hipHostFree(  u_d_h  );
	hipHostFree(  v_d_h  );
	hipHostFree(  w_d_h  );
	hipHostFree(  a_d_h  );
	hipHostFree(   et_h  );
	hipHostFree(   ex_h  );
	hipHostFree(   ey_h  );
	hipHostFree(   ez_h  );
	hipHostFree(   wt_h  );
	hipHostFree( t_c_h );
	hipHostFree( t_m_h );
	hipHostFree( t_b_h );
	hipHostFree( t_p_h );
	hipHostFree( t_u_h );
	hipHostFree( t_v_h );
	hipHostFree( t_w_h );
	hipHostFree( t_g_h );
	hipHostFree( t_h_h );
	if(myid==0){
	hipHostFree( c_f_h );
	hipHostFree( m_f_h );
	hipHostFree( b_f_h );
	hipHostFree( p_f_h );
	hipHostFree( u_f_h );
	hipHostFree( v_f_h );
	hipHostFree( w_f_h );
	hipHostFree( a_f_h );
	hipHostFree( xz_f_h );
	hipHostFree( lx );
	hipHostFree( lz );
	}
	hipHostFree( xz_d_h );

	hipFree( xz_d  );
	hipFree(  c_d  );
	hipFree(  m_d  );
	hipFree(  b_d  );
	hipFree(  p_d  );
	hipFree(  u_d  );
	hipFree(  v_d  );
	hipFree(  w_d  );
	hipFree(  a_d  );
	hipFree(  h  );
	hipFree(  g  );
	hipFree( h_t  );
	hipFree( g_t  );
	hipFree( gra_c );
	hipFree( gra_m );
	hipFree( t_c );
	hipFree( t_m );
	hipFree( t_b );
	hipFree( t_p );
	hipFree( t_u );
	hipFree( t_v );
	hipFree( t_w );
	hipFree( t_g );
	hipFree( t_h );
	hipFree( c );
	hipFree( m );
	hipFree( b );
	hipFree( p );
	hipFree( u );
	hipFree( v );
	hipFree( w );
	hipFree( a );

	MPI_Finalize();
	return 0;
	}
